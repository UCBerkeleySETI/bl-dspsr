#include "hip/hip_runtime.h"
//-*-C++-*-
/***************************************************************************
 *
 *   Copyright (C) 2010 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "dsp/UWBFloatUnpackerCUDA.h"
#include "dsp/Operation.h"
#include "dsp/MemoryCUDA.h"

#include "Error.h"

using namespace std;

void check_error_stream (const char*, hipStream_t);

__global__ void uwb_unpack_fpt_float_kernel (float2 * to, 
                                             const float2 * from, 
                                             uint64_t out_pol_stride,
                                             unsigned nblock,
                                             uint64_t ndat_per_block)
{
  const uint64_t idat = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (idat >= ndat_per_block)
    return;

  const unsigned ipol = blockIdx.z;
  const unsigned npol = gridDim.z;

  // SFPT ordering, S==1, F==1, so PT
  uint64_t idx = (ipol * ndat_per_block) + idat;
  uint64_t odx = (ipol * out_pol_stride) + idat;

  // block_stride is               npol * ndat_per_block
  const uint64_t in_block_stride = npol * ndat_per_block;

  for (unsigned i=0; i<nblock; i++)
  {
    to[odx] = from[idx];

    idx += in_block_stride;
    odx += ndat_per_block;
  }
}

CUDA::UWBFloatUnpackerEngine::UWBFloatUnpackerEngine (hipStream_t _stream)
{
  stream = _stream;
}

void CUDA::UWBFloatUnpackerEngine::setup ()
{
  // determine cuda device properties for block & grid size
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties (&gpu, device);
}

bool CUDA::UWBFloatUnpackerEngine::get_device_supported (dsp::Memory* memory) const
{
  return dynamic_cast< CUDA::DeviceMemory*> ( memory );
}

void CUDA::UWBFloatUnpackerEngine::set_device (dsp::Memory* memory)
{
}

void CUDA::UWBFloatUnpackerEngine::unpack (const dsp::BitSeries * input, dsp::TimeSeries * output)
{
  const uint64_t ndat = input->get_ndat();
  const unsigned ndat_per_block = input->get_loader()->get_resolution();
  const uint64_t nblock = ndat / ndat_per_block;
  const int npol = input->get_npol();
  const int nchan = input->get_nchan();

  if (nchan > 1)
    throw Error(InvalidState, "CUDA::UWBFloatUnpackerEngine::unpack", "Only nchan == 1 supported"); 

#ifdef _DEBUG
  cerr << "CUDA::UWBFloatUnpackerEngine::unpack ndat=" << ndat 
       << " ndat_per_block=" << ndat_per_block << " npol=" << npol 
       << " nchan=" << nchan << " nblock=" << nblock << endl;
#endif

  if (ndat % ndat_per_block)
    throw Error(InvalidState, "CUDA::UWBFloatUnpackerEngine::unpack", "ndat was not divisible by resolution");
  
  unsigned nthreads = 1024; 
  dim3 blocks = dim3 (ndat_per_block / nthreads, nchan, npol);
  if (ndat_per_block % nthreads > 0)
    blocks.x++;

#ifdef _DEBUG
 cerr << "CUDA::UWBFloatUnpackerEngine::unpack nthreads=" << nthreads 
       << " blocks=(" << blocks.x << "," << blocks.y << "," << blocks.z
       << ")" << endl;
#endif

  // use an float2 to handle the re+im parts of the float
  float2 * from = (float2 *) input->get_rawptr();
  float2 * into = (float2 *) output->get_datptr(0, 0);
  uint64_t pol_stride = 0;

  if (npol == 2)
  {
    float2  * into_a = (float2 *) output->get_datptr(0, 0);
    float2  * into_b = (float2 *) output->get_datptr(0, 1);
    pol_stride = uint64_t(into_b - into_a);
  }

  uwb_unpack_fpt_float_kernel<<<blocks,nthreads,0,stream>>> (into, from, pol_stride, nblock, ndat_per_block);

  //float2 * into = (float2 *) output->get_datptr(0, 0);
  //uwb_unpack_fpt_float_kernel<<<blocks,nthreads,0,stream>>> (into, from, nblock, ndat_per_block);
  
  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error_stream ("CUDA::UWBFloatUnpackerEngine::unpack", stream);
}
