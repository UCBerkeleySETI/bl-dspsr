#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2016 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/TimeSeriesCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "Error.h"

void check_error_stream (const char*, hipStream_t);

using namespace std;

template<typename T>
__global__ void copy_data_fpt_kernel(T * to, T * from,
                                     uint64_t to_stride, uint64_t from_stride,
                                     uint64_t ndat)
{
  uint64_t dx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dx >= ndat)
    return;
  to[blockIdx.y * to_stride + dx] = from[blockIdx.y * from_stride + dx];
}

CUDA::TimeSeriesEngine::TimeSeriesEngine (dsp::Memory * _memory)
{
  memory = dynamic_cast<CUDA::DeviceMemory*>(_memory);
  buffer = NULL;
  buffer_size = 0;

  pinned_memory = new CUDA::PinnedMemory;
  host_buffer = NULL;
  host_buffer_size = 0;
}

CUDA::TimeSeriesEngine::~TimeSeriesEngine ()
{
  if (buffer)
    memory->do_free (buffer);
  buffer = 0;
}

void CUDA::TimeSeriesEngine::prepare (dsp::TimeSeries * parent)
{
  to = parent;
}

void CUDA::TimeSeriesEngine::prepare_buffer (unsigned nbytes)
{
  if (nbytes > buffer_size)
  {
    if (buffer)
      memory->do_free (buffer);
    buffer_size = nbytes;
    buffer = memory->do_allocate (buffer_size);
    memory->do_zero(buffer, buffer_size);
  }
}

// copy data from another time series to this time series
void CUDA::TimeSeriesEngine::copy_data_fpt (const dsp::TimeSeries* from, 
    uint64_t idat_start, uint64_t ndat)
{
  nchan = to->get_nchan();
  npol  = to->get_npol();
  ndim  = to->get_ndim();

  // current cuda device that is executing this function
  hipGetDevice (&device);

#ifdef _DEBUG
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt from=" << (void *) from 
       << " idat_start=" << idat_start << " ndat=" << ndat << " device=" << device << endl;
#endif

  // stream and device upon which to TSE exists
  to_stream = memory->get_stream();
  to_device = memory->get_device();

  // stream and device upon which from TSE exists
  const CUDA::DeviceMemory * from_mem = dynamic_cast<const CUDA::DeviceMemory*>( from->get_memory());
  from_stream = from_mem->get_stream();
  from_device = from_mem->get_device();

  if (!from_mem)
    throw Error (FailedSys, "CUDA::TimeSeriesEngine::copy_data_fpt", "From TimeSeries did not use DeviceMemory");

  ichanpol_stride = 0;
  ochanpol_stride = 0;
  bchanpol_stride = ndat;

  if (npol > 1)
  {
    ochanpol_stride = to->get_datptr (0,1) - to->get_datptr (0,0);
    ichanpol_stride = from->get_datptr (0,1) - from->get_datptr (0,0);
  }
  else if (nchan > 1)
  {
    ochanpol_stride = to->get_datptr (1,0) - to->get_datptr (1,0);
    ichanpol_stride = from->get_datptr (1,0) - from->get_datptr (1,0);
  }
  else
  {
    ; 
  }

  ichanpol_stride /= ndim;
  ochanpol_stride /= ndim;

#ifdef _DEBUG
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt streams to="
       << (void*) to_stream << " from=" << (void*) from_stream << endl;
  cerr << "CUDA::TimeSeriesEngine::copy_data_fpt device to=" << device 
       << " from=" << from_device << endl;
  cerr  << "CUDA::TimeSeriesEngine::copy_data_fpt nchan=" << nchan << " ndim=" << ndim << " npol=" << npol << " ndat=" << ndat << endl;
  cerr  << "CUDA::TimeSeriesEngine::copy_data_fpt istride=" << ichanpol_stride << " ostride=" << ochanpol_stride << " bstride=" << bchanpol_stride << endl;
#endif

  // configure the kernels
  nthread = 1024;
  if (nthread > ndat)
    nthread = ndat;
  blocks = dim3 (ndat / nthread, nchan*npol);
  if (ndat % nthread)
    blocks.x++;

#ifdef _DEBUG
  cerr << "blocks=(" << blocks.x << "," << blocks.y << ") threads=" << nthread << endl;
#endif

  if (from_device == to_device)
    if (from_stream == to_stream) 
      copy_data_fpt_same_stream (from, idat_start, ndat);
    else
      copy_data_fpt_same_device (from, idat_start, ndat);
  else
    copy_data_fpt_diff_device (from, idat_start, ndat);
}

// if both time series are within the same stream
void CUDA::TimeSeriesEngine::copy_data_fpt_same_stream (const dsp::TimeSeries* from,
    uint64_t idat_start, uint64_t ndat)
{
  hipStream_t stream = memory->get_stream();
  copy_data_fpt_kernel_multidim (to->get_datptr (0,0), from->get_datptr (0,0),
                                 ochanpol_stride, ichanpol_stride, 
                                 idat_start, ndat, stream);
}

// if both time series are in different streams, but the same device
void CUDA::TimeSeriesEngine::copy_data_fpt_same_device (const dsp::TimeSeries* from,
    uint64_t idat_start, uint64_t ndat)
{
  size_t nbytes = nchan * ndim * npol * ndat * sizeof(float);

  // stream upon which from TSE exists
  CUDA::TimeSeriesEngine * from_engine = dynamic_cast<CUDA::TimeSeriesEngine*>(from->get_engine());
  const CUDA::DeviceMemory * from_mem = dynamic_cast<const CUDA::DeviceMemory*>( from->get_memory());

  // ensure the buffers in each time series are allocated
  from_engine->prepare_buffer (nbytes);
  prepare_buffer (nbytes);

  // copy from -> from_buffer
  copy_data_fpt_kernel_multidim ((float *) from_engine->buffer, from->get_datptr (0,0), 
                                 bchanpol_stride, ichanpol_stride, 
                                 idat_start, ndat, from_stream);

  // copy from_buffer -> to_buffer  
  hipMemcpyAsync (buffer, from_engine->buffer, nbytes, hipMemcpyDeviceToDevice, from_stream);
  hipStreamSynchronize(from_stream);

  // copy buffer -> to
  copy_data_fpt_kernel_multidim (to->get_datptr (0,0), (float *) buffer, 
                                 ochanpol_stride, bchanpol_stride, 
                                 0, ndat, to_stream);
}

// if both time series are in different streams, but the same device
void CUDA::TimeSeriesEngine::copy_data_fpt_diff_device (const dsp::TimeSeries* from,
    uint64_t idat_start, uint64_t ndat)
{
  size_t nbytes = nchan * ndim * npol * ndat * sizeof(float);

  // if the current device is not the to device, switch and allocate
  if (device != to_device)
    hipSetDevice (to_device);
  prepare_buffer (nbytes);

  // switch to the from_device
  hipSetDevice (from_device);

  // ensure buffer is allocated
  CUDA::TimeSeriesEngine * from_engine = dynamic_cast<CUDA::TimeSeriesEngine*>(from->get_engine());
  from_engine->prepare_buffer (nbytes);

  // copy from -> from_buffer
  copy_data_fpt_kernel_multidim ((float *) from_engine->buffer, from->get_datptr (0,0),
                                 bchanpol_stride, ichanpol_stride,
                                 idat_start, ndat, from_stream);

  // if the host buffer is too small, allocate some pinned memory
  if (host_buffer_size < nbytes)
  {
    if (host_buffer)
      pinned_memory->do_free (host_buffer);
    host_buffer = pinned_memory->do_allocate (nbytes);
    host_buffer_size = nbytes;
  }

  // copy from_buffer -> host_buffer
  hipMemcpyAsync (host_buffer, from_engine->buffer, nbytes, hipMemcpyDeviceToHost, from_stream);

  // wait for the D2H transfer to complete before continuing
  hipStreamSynchronize (from_stream);

  // switch to the to_device
  hipSetDevice (to_device);

  // copy host_buffer -> to_buffer
  hipMemcpyAsync (buffer, host_buffer, nbytes, hipMemcpyHostToDevice, to_stream);

  // copy to_buffer -> to
  copy_data_fpt_kernel_multidim (to->get_datptr (0,0), (float *) buffer,
                                 ochanpol_stride, bchanpol_stride,
                                 0, ndat, to_stream);

  if (to_device != device)
    hipSetDevice (device);
}


void CUDA::TimeSeriesEngine::copy_data_fpt_kernel_multidim (float * to, const float * from, 
                                 uint64_t to_stride, uint64_t from_stride, 
                                 uint64_t idat_start, uint64_t ndat, 
                                 hipStream_t stream)
{
  if (ndim == 2)
  {
    float2 * to_ptr   = (float2 *) to;
    float2 * from_ptr = (float2 *) from;
    copy_data_fpt_kernel<float2><<<blocks,nthread,0,stream>>> (
      to_ptr, from_ptr + idat_start, to_stride, from_stride, ndat);
  }
  else
  {
    float * from_ptr = (float *) from;
    copy_data_fpt_kernel<float><<<blocks,nthread,0,stream>>> (
      to, from_ptr + idat_start, to_stride, from_stride, ndat);
  }
}





