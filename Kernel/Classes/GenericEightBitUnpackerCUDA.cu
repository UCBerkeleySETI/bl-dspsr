#include "hip/hip_runtime.h"
//-*-C++-*-
/***************************************************************************
 *
 *   Copyright (C) 2015 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/GenericEightBitUnpackerCUDA.h"
#include "dsp/Operation.h"

#include "Error.h"

using namespace std;

void check_error_stream (const char*, hipStream_t);

/*
 * Simple CUDA 8-bit unpack kernel
 * This kernel is not optimized; in particular, data access is not coalesced.
 */

template<typename T>
__global__ void unpack (unpack_dimensions dim,
			float scale, float offset,
			const T* input,
			float* output, uint64_t output_stride)
{
  uint64_t idat  = blockIdx.x * blockDim.x + threadIdx.x
                 + blockIdx.z * blockDim.x * gridDim.x;
  unsigned ichan = blockIdx.y;
  unsigned ipol  = threadIdx.y;
  unsigned idim  = threadIdx.z;

  unsigned input_stride = dim.nchan * dim.npol * dim.ndim;

  if (idat >= dim.ndat)
    return;

  input += input_stride*idat + dim.ndim * (dim.npol*ichan + ipol) + idim;

  output += output_stride * (dim.npol*ichan + ipol) + dim.ndim*idat + idim;

  *output = (float(*input) + offset) * scale;
}

// defined in FoldCUDA.C
static ostream& operator << (ostream& os, const dim3& c)
{
  return os << "[" << c.x << "," << c.y << "," << c.z << "]";
}

void generic_8bit_unpack (hipStream_t stream, 
			  const unpack_dimensions& dim,
			  const dsp::BitTable* table,
			  const unsigned char* input,
			  float* output, uint64_t stride)
{
  unsigned max_threads_per_block = 256;
  unsigned min_threads_per_block = 32;
  unsigned max_blocks_per_dim = 65535;

  unsigned datum_threads = max_threads_per_block / (dim.npol * dim.ndim);
  if (datum_threads > dim.ndat)
    datum_threads = min_threads_per_block;

  unsigned datum_blocks_x = dim.ndat / datum_threads;
  unsigned datum_blocks_z = 1;

  if (datum_blocks_x > max_blocks_per_dim)
  {
    datum_blocks_z = datum_blocks_x / max_blocks_per_dim;
    if (datum_blocks_x % max_blocks_per_dim)
      datum_blocks_z ++;

    datum_blocks_x /= datum_blocks_z;
  }

  while (dim.ndat > datum_threads * datum_blocks_z * datum_blocks_x)
    datum_blocks_x ++;

  dim3 blockDim (datum_threads, dim.npol, dim.ndim);
  dim3 gridDim (datum_blocks_x, dim.nchan, datum_blocks_z);

  // cerr << "blockDim=" << blockDim << " gridDim=" << gridDim << endl;

  if (table->get_type() == dsp::BitTable::TwosComplement)
  {
    const int8_t* signed_input = reinterpret_cast<const int8_t*> (input);
    float offset = 0.5;
    unpack<<<gridDim,blockDim,0,stream>>> (dim, table->get_scale(), offset, 
					   signed_input, output, stride);
  }
  else if (table->get_type() == dsp::BitTable::OffsetBinary)
  {
    float offset = -127.5;
    unpack<<<gridDim,blockDim,0,stream>>> (dim, table->get_scale(), offset, 
					   input, output, stride);
  }
  else
    throw Error (InvalidState, "generic_8bit_unpack",
		 "unknown BitTable::Type");

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error_stream ("generic_8bit_unpack", stream);
}

