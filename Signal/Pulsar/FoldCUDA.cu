#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2010 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/FoldCUDA.h"
#include "dsp/MemoryCUDA.h"

//#define _DEBUG

#include "Error.h"
#include "debug.h"

#include <hip/hip_complex.h>
#include <memory>

#ifdef __CUDA_ARCH__
    #if (__CUDA_ARCH__ >= 300)
        #define HAVE_SHFL
    #else
        #define NO_SHFL
    #endif
#endif

using namespace std;

CUDA::FoldEngine::FoldEngine (hipStream_t _stream, bool _hits_on_gpu)
{
  use_set_bins = false;
  d_bin = 0;
  d_bin_size = 0;

  binplan = 0;
  binplan_size = 0;
  binplan_nbin = 0;

  stream = _stream;

  d_profiles = new dsp::PhaseSeries;
  d_profiles->set_memory( new CUDA::DeviceMemory(stream) );
  
  hits_on_gpu = _hits_on_gpu;
  if (hits_on_gpu)
    d_profiles->set_hits_memory (new CUDA::DeviceMemory(stream) );

  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::FoldEngine hits_on_gpu=" << hits_on_gpu << endl;

  // no data on either the host or device
  synchronized = true;

}

CUDA::FoldEngine::~FoldEngine ()
{
  if (d_bin)
    hipFree (d_bin);
}

void CUDA::FoldEngine::set_nbin (unsigned nbin)
{
  current_bin = folding_nbin = nbin;
  current_hits = 0;
  ndat_fold = 0;
  binplan_nbin = 0;
}

void CUDA::FoldEngine::set_ndat (uint64_t ndat, uint64_t idat_start)
{
  if (ndat > binplan_size)
  {
    if (binplan)
      hipHostFree (binplan);

    hipHostMalloc ((void**)&binplan, ndat * sizeof(bin));
    binplan_size = ndat;
  }
}

void CUDA::FoldEngine::set_bin (uint64_t idat, double d_ibin, 
        double bins_per_sample)
{
  unsigned ibin = unsigned (d_ibin);
  if (ibin != current_bin)
  {
    /* store the number of time samples to integrate
       in the interval that just ended */
    if (binplan_nbin)
      binplan[binplan_nbin-1].hits = current_hits;

    bin start;
    start.offset = idat;
    start.ibin = ibin;

    if (binplan_nbin >= binplan_size)
      throw Error (InvalidState, "CUDA::FoldEngine::set_bin",
                   "binplan nbin=%u >= size=%u", binplan_nbin, binplan_size);

    /* start a new interval */
    binplan[binplan_nbin] = start;

    binplan_nbin ++;
    current_bin = ibin;
    current_hits = 0;
  }

  ndat_fold ++;
  current_hits ++;
}

uint64_t CUDA::FoldEngine::get_bin_hits (int ibin){
  return 0; // Fix this
}
uint64_t CUDA::FoldEngine::set_bins (double phi, double phase_per_sample, uint64_t _ndat, uint64_t idat_start)
{
  return 0;
}
dsp::PhaseSeries* CUDA::FoldEngine::get_profiles ()
{
  return d_profiles;
}

void CUDA::FoldEngine::synch (dsp::PhaseSeries* output) try
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::synch this=" << this << endl;

  if (synchronized)
    return;

  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::synch output=" << output << endl;

  if (!transfer)
    transfer = new dsp::TransferPhaseSeriesCUDA(stream);

  transfer->set_kind( hipMemcpyDeviceToHost );
  transfer->set_input( d_profiles );
  transfer->set_output( output );
  transfer->set_transfer_hits( hits_on_gpu );
  transfer->operate ();

  synchronized = true;
}
catch (Error& error)
{
  throw error += "CUDA::FoldEngine::synch";
}

void CUDA::FoldEngine::send_binplan ()
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::send_binplan ndat=" << ndat_fold 
         << " intervals=" << binplan_nbin << endl;

  if (binplan_nbin == 0)
    return;

  if (current_hits)
    binplan[binplan_nbin-1].hits = current_hits;

  current_hits = 0;

  if (dsp::Operation::verbose)
    cerr << "CUDA::FoldEngine::send_binplan"
            " first=" << binplan[0].ibin << 
            " last=" << binplan[binplan_nbin-1].ibin <<
            " stream=" << stream << endl;

  uint64_t mem_size = binplan_nbin * sizeof(bin);

  if (binplan_nbin > d_bin_size)
  {
    if (d_bin)
      hipFree (d_bin);

    hipMalloc ((void**)&d_bin, mem_size);
    d_bin_size = binplan_nbin;
  }

  // copy the kernel accross
  hipError_t error;

  if (stream)
    error = hipMemcpyAsync (d_bin, binplan, mem_size,
                             hipMemcpyHostToDevice, stream);
  else
    error = hipMemcpy (d_bin, binplan, mem_size, hipMemcpyHostToDevice);

  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::FoldEngine::set_binplan",
                 "hipMemcpy%s %s", 
                 stream?"Async":"", hipGetErrorString (error));
}

/* All CUDA folding kernels utilise the dimensionality:
 *   ipol = blockIdx.z
 *   npol = gridDim.z
 *   ichan = blockIdx.y
 *   nchan = gridDim.y
 */

// 2dim data
__global__ void fold1bin2dim (const hipFloatComplex * in_base,
           unsigned in_span,
           hipFloatComplex * out_base,
           unsigned out_span,
           unsigned nbin,
           unsigned binplan_size,
           const CUDA::bin* binplan)
{
  in_base  += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base += out_span * (blockIdx.y * gridDim.z + blockIdx.z);

  for (unsigned ibin=threadIdx.x; ibin<binplan_size; ibin+=blockDim.x)
  {
    const hipFloatComplex * input = in_base + binplan[ibin].offset;
    hipFloatComplex total = make_hipComplex (0,0);
    for (unsigned i=0; i < binplan[ibin].hits; i++)
      total = hipCaddf (total, input[i]);
    const unsigned output_ibin = binplan[ibin].ibin;
    atomicAdd(&(out_base[output_ibin].x), total.x);
    atomicAdd(&(out_base[output_ibin].y), total.y);
  }
}

__global__ void fold1bin2dim_shared (const hipFloatComplex * in_base,
           unsigned in_span,
           hipFloatComplex * out_base,
           unsigned out_span,
           unsigned nbin,
           unsigned binplan_size,
           const CUDA::bin* binplan)
{
  // one shared memory bin for each output phase bin for this chanpol
  extern __shared__ hipFloatComplex f1b2d_shared[];

  // pointers for the current channel and polarisation
  in_base  += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base += out_span * (blockIdx.y * gridDim.z + blockIdx.z);

  // coalesced read the existing phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
    f1b2d_shared[ibin] = out_base[ibin];

  __syncthreads();

  for (unsigned ibin=threadIdx.x; ibin<binplan_size; ibin+=blockDim.x)
  {
    // input pointer for this phase bin
    const hipFloatComplex * input = in_base + binplan[ibin].offset;
    hipFloatComplex total = make_hipComplex (0,0);
    for (unsigned i=0; i < binplan[ibin].hits; i++)
      total = hipCaddf (total, input[i]);
    const unsigned output_ibin = binplan[ibin].ibin;
    atomicAdd(&(f1b2d_shared[output_ibin].x), total.x);
    atomicAdd(&(f1b2d_shared[output_ibin].y), total.y);
  }

  __syncthreads();

  // coalesced write the new phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
    out_base[ibin] = f1b2d_shared[ibin];
}

// each warp will fold a single binplan bin
__global__ void fold1bin2dim_warp (const float2* in_base,
           unsigned in_span,
           float2* out_base,
           unsigned out_span,
           unsigned nbin,
           unsigned binplan_size,
           CUDA::bin* binplan)
{
  extern __shared__ hipFloatComplex warp_fold[];

  const int warps_per_block = blockDim.x / 32;
  const int warp_idx = threadIdx.x & 0x1F;      // % 32
  const int warp_num = threadIdx.x / 32;
  
  // the ibin that threads in this warp will add up together
  const int ibin = blockIdx.x * warps_per_block + warp_num;

  hipFloatComplex total = make_hipComplex (0,0);

  // only add up bins that we have
  if (ibin < binplan_size)
  {
    in_base += in_span * (blockIdx.y * gridDim.z + blockIdx.z);

    // start/end sample for this input bin
    const int sbin = binplan[ibin].offset;
    const int ebin = sbin + binplan[ibin].hits;

    // each thread of a warp will load samples for this ibin
    for (int i=sbin+warp_idx; i<ebin; i+=32)
    {
      total = hipCaddf (total, in_base[i]);
    }
  }

  // now add totals together
#ifdef HAVE_SHFL
  total.x += __shfl_down (total.x, 16);
  total.x += __shfl_down (total.x, 8);
  total.x += __shfl_down (total.x, 4);
  total.x += __shfl_down (total.x, 2);
  total.x += __shfl_down (total.x, 1);

  total.y += __shfl_down (total.y, 16);
  total.y += __shfl_down (total.y, 8);
  total.y += __shfl_down (total.y, 4);
  total.y += __shfl_down (total.y, 2);
  total.y += __shfl_down (total.y, 1);

  // copy to shm for warp 0 to write out to gmem
  if (warp_idx == 0)
    warp_fold[warp_num] = total; 
  __syncthreads();

  if (warp_num == 0)
  {
    out_base += out_span * (blockIdx.y * gridDim.z + blockIdx.z);
    const int ibin = blockIdx.x * warps_per_block + warp_idx;
    if (ibin >= binplan_size)
      return;
    int output_ibin = binplan[ibin].ibin;
    out_base[ output_ibin ] = hipCaddf (out_base[ output_ibin ], warp_fold[warp_idx]);
  }
#endif
#ifdef NO_SHFL
  int last_offset = 16;
  warp_fold[threadIdx.x] = total;
  __syncthreads();
  for (int offset = last_offset; offset > 0;  offset >>= 1)
  {
    if (warp_idx < offset)
      warp_fold[threadIdx.x] = hipCaddf(warp_fold[threadIdx.x], warp_fold[threadIdx.x + offset]);
    __syncthreads();
  }

  if (warp_idx == 0)
  {
    if (ibin < binplan_size)
    {
      out_base += out_span * (blockIdx.y * gridDim.z + blockIdx.z);
      int output_ibin = binplan[ibin].ibin;
      out_base[ output_ibin ] = hipCaddf (out_base[ output_ibin ], warp_fold[threadIdx.x]);
    }
  }
#endif
}


// 1dim kernels 
__global__ void fold1bin1dim_shared (const float* in_base, unsigned in_span,
           float* out_base, unsigned out_span, 
           unsigned nbin, unsigned binplan_size,
           CUDA::bin* binplan)
{
  // one shared memory bin for each output phase bin for this chanpol
  extern __shared__ float f1b1d_shared[];

  // pointers for the current channel and polarisation
  in_base  += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base += out_span * (blockIdx.y * gridDim.z + blockIdx.z);

  // coalesced read the existing phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
    f1b1d_shared[ibin] = out_base[ibin];

  __syncthreads();

  for (unsigned ibin=threadIdx.x; ibin<binplan_size; ibin+=blockDim.x)
  {
    // input pointer for this phase bin
    const float * input = in_base + binplan[ibin].offset;
    float total = 0;
    for (unsigned i=0; i < binplan[ibin].hits; i++)
      total += input[i];
    const unsigned obin = binplan[ibin].ibin;
    atomicAdd(&(f1b1d_shared[obin]), total);
  }

  __syncthreads();

  // coalesced write all the new phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
    out_base[ibin] = f1b1d_shared[ibin];
}

__global__ void fold1bin1dim (const float* in_base, unsigned in_span,
           float* out_base, unsigned out_span,
           unsigned nbin, unsigned binplan_size,
           CUDA::bin* binplan)
{
  in_base  += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base += out_span * (blockIdx.y * gridDim.z + blockIdx.z);

  for (unsigned ibin=threadIdx.x; ibin<binplan_size; ibin+=blockDim.x)
  {
    const float* input = in_base + binplan[ibin].offset;
    float total = 0;
    for (unsigned i=0; i < binplan[ibin].hits; i++)
      total += input[i];
    const unsigned obin = binplan[ibin].ibin;
    atomicAdd(&(out_base[obin]), total);
  }
}

// 2dim kernel hits
__global__ void fold1bin2dimhits_shared (const float2* in_base, unsigned in_span,
           float2* out_base, unsigned out_span,
           unsigned* hits_base,
           unsigned nbin, unsigned binplan_size, CUDA::bin* binplan)
{
  // one shared memory bin for each output phase bin for this chanpol
  extern __shared__ float2 f1b2dh_shared[];
  unsigned * hits_shared = (unsigned *) (f1b2dh_shared + nbin);

  //           stride   * (  ichan    *  npol      + ipol     )
  in_base   += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base  += out_span * (blockIdx.y * gridDim.z + blockIdx.z);
  hits_base += nbin * blockIdx.y;

  // coalesced read the existing phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
  {
    f1b2dh_shared[ibin] = out_base[ibin];
    if (blockIdx.z == 0)
      hits_shared[ibin] = hits_base[ibin];
  }
  __syncthreads();

  for (unsigned ibin=threadIdx.x; ibin < binplan_size; ibin += blockDim.x)
  {
    const float2* input = in_base + binplan[ibin].offset;
    float2 total = make_hipComplex (0,0);
    unsigned hits = 0;
    for (unsigned i=0; i < binplan[ibin].hits; i++)
    {
      total = hipCaddf( total, input[i]);
      if (blockIdx.z == 0)
        hits += (input[i].x != 0);
    }
    const unsigned obin = binplan[ibin].ibin;
    atomicAdd(&(f1b2dh_shared[obin].x), total.x);
    atomicAdd(&(f1b2dh_shared[obin].y), total.y);
    if (blockIdx.z == 0)
      atomicAdd(&(hits_shared[obin]), hits);
  }

  __syncthreads();

  // coalesced write all the new phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
  {
    out_base[ibin] = f1b2dh_shared[ibin];
    if (blockIdx.z == 0)
      hits_base[ibin] = hits_shared[ibin];
  }
}

__global__ void fold1bin2dimhits (const float2* in_base, unsigned in_span,
           float2* out_base, unsigned out_span,
           unsigned* hits_base,
           unsigned nbin, unsigned binplan_size, CUDA::bin* binplan)
{
  //           stride   * (  ichan    *  npol      + ipol     )
  in_base   += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base  += out_span * (blockIdx.y * gridDim.z + blockIdx.z);
  hits_base += nbin * blockIdx.y;

  for (unsigned ibin=threadIdx.x; ibin < binplan_size; ibin += blockDim.x)
  {
    const float2* input = in_base + binplan[ibin].offset;
    float2 total = make_hipComplex (0,0);
    unsigned hits = 0;
    for (unsigned i=0; i < binplan[ibin].hits; i++)
    {
      total = hipCaddf( total, input[i]);
      if (blockIdx.z == 0)
        hits += (input[i].x != 0);
    }
    const unsigned obin = binplan[ibin].ibin;
    atomicAdd(&(out_base[obin].x), total.x);
    atomicAdd(&(out_base[obin].y), total.y);
    if (blockIdx.z == 0)
      atomicAdd(&(hits_base[obin]), hits);
  }
}

__global__ void fold1bin1dimhits_shared (const float* in_base, unsigned in_span,
           float* out_base, unsigned out_span,
           unsigned* hits_base,
           unsigned nbin, unsigned binplan_size, CUDA::bin* binplan)
{
  // one shared memory bin for each output phase bin for this chanpol
  extern __shared__ float f1b1dh_shared[];
  unsigned * hits_shared = (unsigned *) (f1b1dh_shared + nbin);

  //           stride   * (  ichan    *  npol      + ipol     )
  in_base   += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base  += out_span * (blockIdx.y * gridDim.z + blockIdx.z);
  hits_base += nbin * blockIdx.y;

  // coalesced read the existing phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
  {
    f1b1dh_shared[ibin] = out_base[ibin];
    if (blockIdx.z == 0)
      hits_shared[ibin] = hits_base[ibin];
  }
  __syncthreads();

  for (unsigned ibin=threadIdx.x; ibin < binplan_size; ibin += blockDim.x)
  {
    const float* input = in_base + binplan[ibin].offset;
    float total = 0;
    unsigned hits = 0;
    for (unsigned i=0; i < binplan[ibin].hits; i++)
    {
      total += input[i];
      if (blockIdx.z == 0)
        hits += (input[i] != 0);
    }
    const unsigned obin = binplan[ibin].ibin;
    atomicAdd(&(f1b1dh_shared[obin]), total);
    // for ipol == 0 only
    if (blockIdx.z == 0)
      atomicAdd(&(hits_shared[obin]), hits);
  }

  __syncthreads();

  // coalesced write all the new phase bin values
  for (unsigned ibin=threadIdx.x; ibin<nbin; ibin+=blockDim.x)
  {
    out_base[ibin] = f1b1dh_shared[ibin];
    if (blockIdx.z == 0)
      hits_base[ibin] = hits_shared[ibin];
  }
}

__global__ void fold1bin1dimhits (const float* in_base, unsigned in_span,
			     float* out_base, unsigned out_span,
           unsigned* hits_base,
			     unsigned nbin, unsigned binplan_size, CUDA::bin* binplan)
{
  //           stride   * (  ichan    *  npol      + ipol     )
  in_base   += in_span  * (blockIdx.y * gridDim.z + blockIdx.z);
  out_base  += out_span * (blockIdx.y * gridDim.z + blockIdx.z);
  hits_base += nbin * blockIdx.y;
  
  for (unsigned ibin=threadIdx.x; ibin < binplan_size; ibin += blockDim.x)
  {
    const float* input = in_base + binplan[ibin].offset;
    float total = 0;
    unsigned hits = 0;
    for (unsigned i=0; i < binplan[ibin].hits; i++)
    {
      total += input[i];
      hits += (input[i] != 0);
    }
    const unsigned obin = binplan[ibin].ibin;
    atomicAdd(&(out_base[obin]), total);
    // ipol == 0 only
    if (blockIdx.z == 0)
      atomicAdd(&(hits_base[obin]), hits);
  }
}

std::ostream& operator<< (std::ostream& ostr, const dim3& v)
{
  return ostr << "(" << v.x << "," << v.y << "," << v.z << ")";
}

void check_error (const char*);
void check_error_stream (const char*, hipStream_t);

void CUDA::FoldEngine::fold ()
{
  setup ();
  send_binplan ();

  // total number of input phase bins to be opereated (capped at folding_nbin)
  unsigned bin_dim = folding_nbin;
  if (binplan_nbin < folding_nbin)
    bin_dim = binplan_nbin;

  // number of threads in the block (capped a max TPB)
  unsigned bin_threads = bin_dim;
  if (bin_threads > 1024);
    bin_threads = 1024;

  // to ensure block coherrency
  unsigned bin_blocks = 1;

  dim3 blockDim (bin_threads, 1, 1);
  dim3 gridDim (bin_blocks, nchan, npol);

#if 0
  cerr << "bin_dim=" << bin_dim << endl;
  cerr << "blockDim=" << blockDim << endl;
  cerr << "gridDim=" << gridDim << endl;
#endif

  DEBUG("bin_dim=" << bin_dim);
  DEBUG("bin_threads=" << bin_threads << " bin_blocks=" << bin_blocks);
  DEBUG("input=" << (void *) input << " output=" << (void *) output);
  DEBUG("input span=" << input_span << " output span=" << output_span);
  DEBUG("ndim=" << ndim << " nbin=" << folding_nbin << " binplan_nbin=" << binplan_nbin);
  DEBUG("hits_on_gpu=" << hits_on_gpu << " zeroed_samples=" << zeroed_samples << " hits_nchan=" << hits_nchan);

  size_t shared_max = 32768;
  size_t shared_bytes = folding_nbin * sizeof(float) * ndim;
  if (hits_on_gpu && zeroed_samples && hits_nchan == nchan)
  {
    shared_bytes += folding_nbin * sizeof(unsigned);
    if (ndim == 2)
    {
      if (shared_bytes <= shared_max)
        fold1bin2dimhits_shared<<<gridDim,blockDim,shared_bytes,stream>>> ((float2*)input, input_span/2,
                   (float2*) output, output_span/2, hits,
                   folding_nbin, binplan_nbin, d_bin);
      else
        fold1bin2dimhits<<<gridDim,blockDim,0,stream>>> ((float2*)input, input_span/2,
                   (float2*) output, output_span/2, hits,
                   folding_nbin, binplan_nbin, d_bin);
    }
    else
    {
      if (shared_bytes <= shared_max)
        fold1bin1dimhits_shared<<<gridDim,blockDim,shared_bytes,stream>>> (input, input_span,
                   output, output_span, hits,
                   folding_nbin, binplan_nbin, d_bin);
      else
        fold1bin1dimhits<<<gridDim,blockDim,0,stream>>> (input, input_span,
                   output, output_span, hits,
                   folding_nbin, binplan_nbin, d_bin);
    }
  }
  else
  {
    if (ndim == 2)
    {
      if (shared_bytes <= shared_max)
      {
        fold1bin2dim_shared<<<gridDim,blockDim,shared_bytes,stream>>> ((hipFloatComplex *) input, input_span/2,
                 (hipFloatComplex *) output, output_span/2,
                 folding_nbin, binplan_nbin, d_bin);
      }
      else
      {
        fold1bin2dim<<<gridDim,blockDim,0,stream>>> ((hipFloatComplex *) input, input_span/2,
                   (hipFloatComplex *) output, output_span/2,
                   folding_nbin, binplan_nbin, d_bin);
      }
/*
      dim3 threads(1024, 1, 1);
      unsigned nwarps = threads.x / 32;
      dim3 blocks (binplan_nbin/nwarps, nchan, npol);
      if (binplan_nbin % nwarps)
        blocks.x++;
      size_t sbytes = threads.x * sizeof(float2);
      fold1bin2dim_warp<<<blocks,threads,sbytes,stream>>> ((hipFloatComplex *) input, input_span/2,
                 (hipFloatComplex *) output, output_span/2,
                 folding_nbin, binplan_nbin, d_bin);
*/
    }
    else
    {
      if (shared_bytes <= shared_max)
        fold1bin1dim_shared<<<gridDim,blockDim,shared_bytes,stream>>> (input, input_span,
                   output, output_span,
                   folding_nbin, binplan_nbin, d_bin);
      else 
        fold1bin1dim<<<gridDim,blockDim,0,stream>>> (input, input_span,
                   output, output_span,
                   folding_nbin, binplan_nbin, d_bin);
    }
  }

  // profile on the device is no longer synchronized with the one on the host
  synchronized = false;

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    if (stream)
      check_error_stream ("CUDA::FoldEngine::fold", stream);
    else
      check_error ("CUDA::FoldEngine::fold");
}

