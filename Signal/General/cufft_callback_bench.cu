#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2015 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#if HAVE_CONFIG_H
#include <config.h>
#endif

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_fp16.h>
#include <hipfft/hipfftXt.h>

#include "CUFFTError.h"
#include "CommandLine.h"
#include "RealTimer.h"

#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ void k_unpack (hipFloatComplex * output, const __restrict__ char2 * input, const float scale)
{
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;

  char2 element = input[i];
  output[i] = make_hipComplex ((float) element.x/scale, (float) element.y/scale);
}

__global__ void k_multiply (float2* d_fft, const __restrict__ float2 * kernel, unsigned npart)
{
  const unsigned npt = blockDim.x * gridDim.x;
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;

  // load the kernel for this fine channel
  const float2 k = kernel[i];

  while (i < npt * npart)
  {
    const float2 d = d_fft[i];
    const float x = d.x * k.x - d.y * k.y;
    d_fft[i].y = d.x * k.y + d.y * k.x;
    d_fft[i].x = x;
    i += npt;
  }
}

/////////////////////////////////////////////////////////////////////////
//
// store with multiplication by dedispersion kernel
//
__device__ void CB_convolve_and_storeC (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  // the dedispersion kernel value for this element of the FFT
  const hipfftComplex k = ((hipfftComplex *) callerInfo)[offset];
  ((hipfftComplex*)dataOut)[offset] = hipCmulf (d, k);
}

__device__ hipfftCallbackStoreC d_store_fwd_C = CB_convolve_and_storeC;


/////////////////////////////////////////////////////////////////////////
//
// convert an 8bit number to 32 bit
//
__device__ hipfftComplex cufft_callback_load_8bit(
    void *dataIn, 
    size_t offset, 
    void *callerInfo, 
    void *sharedPtr) 
{
  const __restrict__ char2 in = ((char2 *)dataIn)[offset];
  const float scale = 127.0f;
  return make_hipComplex ((float)in.x/scale, (float) in.y/scale);
  //char2 in = ((char2*)dataIn)[offset];
  //float2 out;
  //out.x = (float) in.x / scale;
  //out.y = (float) in.y / scale;

  //return out;
  //return make_hipComplex ((float) element.x, (float) element.y);
  //return make_hipComplex ((float) element.x/scale, (float) element.y/scale);
}
__device__ hipfftCallbackLoadC d_load_8bit_fwd_C = cufft_callback_load_8bit;


/////////////////////////////////////////////////////////////////////////
//
// convert an 16bit number to 32 bit
//
__device__ hipfftComplex cufft_callback_load_half2(
    void *dataIn,
    size_t offset,
    void *callerInfo,
    void *sharedPtr)
{
  half * ptr = (half*) dataIn + (2*offset);
  return make_hipComplex ( __half2float(ptr[0]), __half2float(ptr[1]));
}

__device__ hipfftCallbackLoadC d_load_half2_fwd_C = cufft_callback_load_half2;


/////////////////////////////////////////////////////////////////////////
//
// store with output filtering on
//
__device__ void CB_filtered_store (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  unsigned nfilt_pos = ((unsigned *) callerInfo)[0];
  unsigned nsamp_filt = ((unsigned *) callerInfo)[1];

  offset -= nfilt_pos;
  if ((offset > 0) && (offset < nsamp_filt))
    ((hipfftComplex*)dataOut)[offset] = d;
}

__device__ hipfftCallbackStoreC d_store_bwd_C = CB_filtered_store;

class Speed : public Reference::Able
{
public:

  Speed ();

  // parse command line options
  void parseOptions (int argc, char** argv);

  // run the test
  void runTest ();

protected:

  int npt;
  int niter;
  unsigned gpu_id;
  bool cuda;
};


Speed::Speed ()
{
  gpu_id = 0;
  niter = 16;
  npt = 1024;
  cuda = false;
}

int main(int argc, char** argv) try
{
  Speed speed;
  speed.parseOptions (argc, argv);
  speed.runTest ();
  return 0;
}
catch (Error& error)
{
  cerr << error << endl;
  return -1;
}

void Speed::parseOptions (int argc, char** argv)
{
  CommandLine::Menu menu;
  CommandLine::Argument* arg;

  menu.set_help_header ("undersampling_speed - measure under sampling speed");
  menu.set_version ("undersampling_speed version 1.0");

  arg = menu.add (npt, 'n', "npt");
  arg->set_help ("number of points in each FFT");

#if HAVE_CUFFT
  arg = menu.add (gpu_id, 'd');
  arg->set_help ("GPU device ID");
#endif

  arg = menu.add (niter, 't', "ninter");
  arg->set_help ("number of iterations (batch/loops)");

#if HAVE_CUFFT
  arg = menu.add (cuda, "cuda");
  arg->set_help ("benchmark CUDA");
#endif

  menu.parse (argc, argv);
}

#if HAVE_CUFFT
void check_error_stream (const char*, hipStream_t);
#endif

void Speed::runTest ()
{
#ifdef _DEBUG
  dsp::Operation::verbose = true;
  dsp::Observation::verbose = true;
#endif

  // assume complex FFTs
  const unsigned ndim = 2;
 
  hipStream_t stream = 0;
  if (cuda)
  {
    cerr << "using GPU " << gpu_id << endl;
    hipError_t err = hipSetDevice(gpu_id); 
    if (err != hipSuccess)
      throw Error (InvalidState, "undersampling_speed",
                   "hipSetDevice failed: %s", hipGetErrorString(err));

    err = hipStreamCreate( &stream );
    if (err != hipSuccess)
      throw Error (InvalidState, "undersampling_speed",
                   "hipStreamCreate failed: %s", hipGetErrorString(err));

  }

  const unsigned ndat = npt * niter;
  const unsigned raw_size = ndat * ndim * sizeof(int8_t);
  const unsigned half2_size = ndat * ndim * sizeof(half);
  const unsigned unpacked_size = ndat * ndim * sizeof(float);
  const unsigned kernel_size = npt * sizeof (hipFloatComplex);

  char2 * raw;
  half2 * input_h2;
  hipfftComplex * input;
  hipfftComplex * buffer;
  hipfftComplex * output;
  hipfftComplex * d_kernel;
  unsigned * d_offsets;
  hipfftResult result;
  size_t work_size;

  hipMalloc ((void **) &raw, raw_size);
  hipMalloc ((void **) &input_h2, half2_size);
  hipMalloc ((void **) &input, unpacked_size);
  hipMalloc ((void **) &buffer, unpacked_size);
  hipMalloc ((void **) &output, unpacked_size);
  hipMalloc ((void **) &d_kernel, kernel_size);
  hipMalloc ((void **) &d_offsets, 2 * sizeof(unsigned));

  hipMemsetAsync ((void *) raw, 0, raw_size, stream);
  hipMemsetAsync ((void *) input, 0, unpacked_size, stream);
  hipMemsetAsync ((void *) input_h2, 0, half2_size, stream);
  hipMemsetAsync ((void *) d_kernel, 0, kernel_size, stream);

  unsigned * h_offsets;
  hipHostMalloc((void **) &h_offsets, 2 * sizeof(unsigned));
  h_offsets[0] = (unsigned) (npt / 15);
  h_offsets[1] = (unsigned) (npt / 15);

  hipMemcpyAsync ((void *) d_offsets, (void *) h_offsets, 2 * sizeof(unsigned), hipMemcpyHostToDevice, stream);

  // all plans are using batched FFTs to ensure at least 1M points

  hipfftHandle plan_batch;
  result = hipfftCreate (&plan_batch);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftCreate(plan_batch)");

  int rank = 1;
  result = hipfftMakePlanMany (plan_batch, rank, &npt, NULL, 0, 0, NULL, 0, 0, 
                              HIPFFT_C2C, niter, &work_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftMakePlanMany (plan_batch)");

  result = hipfftSetStream (plan_batch, stream);
  if (result != HIPFFT_SUCCESS)
    CUFFTError (result, "Speed::runTest", "hipfftSetStream (plan_batch)");


  hipfftHandle plan_callback;
  result = hipfftCreate (&plan_callback);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftCreate(plan_callback)");

  result = hipfftMakePlanMany (plan_callback, rank, &npt, NULL, 0, 0, NULL, 0, 0,
                              HIPFFT_C2C, niter, &work_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftMakePlanMany (plan_callback)");

  result = hipfftSetStream (plan_callback, stream);
  if (result != HIPFFT_SUCCESS)
    CUFFTError (result, "Speed::runTest", "hipfftSetStream (plan_callback)");

  hipfftHandle plan_half;
  result = hipfftCreate (&plan_half);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftCreate(plan_half)");

  result = hipfftMakePlanMany (plan_half, rank, &npt, NULL, 0, 0, NULL, 0, 0,
                              HIPFFT_C2C, niter, &work_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftMakePlanMany (plan_half)");

  result = hipfftSetStream (plan_half, stream);
  if (result != HIPFFT_SUCCESS)
    CUFFTError (result, "Speed::runTest", "hipfftSetStream (plan_half)");

  hipfftHandle plan_bwd;
  result = hipfftCreate (&plan_bwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftCreate(plan_bwd)");

  result = hipfftMakePlanMany (plan_bwd, rank, &npt, NULL, 0, 0, NULL, 0, 0,
                              HIPFFT_C2C, niter, &work_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftMakePlanMany (plan_bwd)");

  result = hipfftSetStream (plan_bwd, stream);
  if (result != HIPFFT_SUCCESS)
    CUFFTError (result, "Speed::runTest", "hipfftSetStream (plan_bwd)");

  hipfftHandle plan_bwd_cb;
  result = hipfftCreate (&plan_bwd_cb);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftCreate(plan_bwd_cb)");
  
  result = hipfftMakePlanMany (plan_bwd_cb, rank, &npt, NULL, 0, 0, NULL, 0, 0,
                              HIPFFT_C2C, niter, &work_size);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftMakePlanMany (plan_bwd_cb)");

  result = hipfftSetStream (plan_bwd_cb, stream);
  if (result != HIPFFT_SUCCESS)
    CUFFTError (result, "Speed::runTest", "hipfftSetStream (plan_bwd_cb)");




  RealTimer timer_batch;
  RealTimer timer_callback;
  RealTimer timer_half;
  RealTimer timer_;
 
  hipfftCallbackLoadC  h_load_8bit_fwd_C;
  hipfftCallbackLoadC  h_load_half2_fwd_C;
  hipfftCallbackStoreC h_store_fwd_C;
  hipfftCallbackStoreC h_store_bwd_C;
  hipError_t error;

  error = hipMemcpyFromSymbolAsync(&h_load_8bit_fwd_C,
                                    HIP_SYMBOL(d_load_8bit_fwd_C),
                                    sizeof(h_load_8bit_fwd_C),
                                    0,
                                    hipMemcpyDeviceToHost,
                                    stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "Speed::runTest",
                 "hipMemcpyFromSymbolAsync failed for h_load_8bit_fwd_C");


  error = hipMemcpyFromSymbolAsync(&h_load_half2_fwd_C,
                                    HIP_SYMBOL(d_load_half2_fwd_C),
                                    sizeof(h_load_half2_fwd_C),
                                    0,
                                    hipMemcpyDeviceToHost,
                                    stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "Speed::runTest",
                 "hipMemcpyFromSymbolAsync failed for h_load_half2_fwd_C");

  error = hipMemcpyFromSymbolAsync(&h_store_fwd_C,
                                    HIP_SYMBOL(d_store_fwd_C),
                                    sizeof(h_store_fwd_C),
                                    0,
                                    hipMemcpyDeviceToHost,
                                    stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "Speed::runTest",
                 "hipMemcpyFromSymbolAsync failed for h_store_fwd_C");

  error = hipMemcpyFromSymbolAsync(&h_store_bwd_C,
                                    HIP_SYMBOL(d_store_bwd_C),
                                    sizeof(h_store_bwd_C),
                                    0,
                                    hipMemcpyDeviceToHost,
                                    stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "Speed::runTest",
                 "hipMemcpyFromSymbolAsync failed for h_store_bwd_C");

  result = hipfftXtSetCallback (plan_callback,
                               (void **)&h_load_8bit_fwd_C,
                               HIPFFT_CB_LD_COMPLEX,
                               0);
  if (result == CUFFT_LICENSE_ERROR)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks", 
                      "CUFFT Callback invalid license");
  cerr << "result=" << result << endl;
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
      "hipfftXtSetCallback (plan_fwd, h_load_8bit_fwd_C)");

/*
  result = hipfftXtSetCallback (plan_half,
                               (void **)&h_load_half2_fwd_C,
                               HIPFFT_CB_LD_COMPLEX,
                               0);
  if (result == CUFFT_LICENSE_ERROR)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
                      "CUFFT Callback invalid license");
  cerr << "result=" << result << endl;
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
      "hipfftXtSetCallback (plan_fwd, h_load_half2_fwd_C)");
*/

  result = hipfftXtSetCallback (plan_callback,
                               (void **)&h_store_fwd_C,
                               HIPFFT_CB_ST_COMPLEX,
                               (void **)&d_kernel);
  if (result == CUFFT_LICENSE_ERROR)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
                      "CUFFT Callback invalid license");
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
      "hipfftXtSetCallback (plan_fwd, h_store_fwd_C)");

  result = hipfftXtSetCallback (plan_bwd_cb,
                               (void **)&h_store_bwd_C,
                               HIPFFT_CB_ST_COMPLEX,
                               (void **)&d_offsets);
  if (result == CUFFT_LICENSE_ERROR)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
                      "CUFFT Callback invalid license");
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
      "hipfftXtSetCallback (plan_bwd_cb, h_store_bwd_C)");


  hipStreamSynchronize (stream);
/*
  timer_half.start();

  result = hipfftExecC2C (plan_half, (hipfftComplex *) input_h2, output, HIPFFT_FORWARD);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftExecC2C(plan_half)");
  hipStreamSynchronize(stream);

  timer_half.stop();
*/

  timer_callback.start ();

  result = hipfftExecC2C (plan_callback, (hipFloatComplex *) raw, buffer, HIPFFT_FORWARD);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftExecC2C(plan_callback)");

  result = hipfftExecC2C (plan_bwd_cb, output, buffer, HIPFFT_BACKWARD);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftExecC2C(plan_callback)");

  hipStreamSynchronize(stream);

  timer_callback.stop ();
  double total_time, time_per_fft, time_us;

  total_time = timer_callback.get_elapsed();
  time_per_fft = total_time / niter;
  time_us = time_per_fft * 1e6;
  cerr << "CALLBACK: total_time=" << total_time << " time_per_fft=" << time_per_fft 
       << " time_us=" << time_us << endl;

  timer_batch.start ();

  unsigned nthreads = 1024;
  unsigned nblocks = ndat / nthreads;
  if (ndat % nthreads != 0)
    nblocks++;

  k_unpack<<<nblocks,nthreads,0,stream>>> (input, raw, 127.0f);

  result = hipfftExecC2C (plan_batch, input, buffer, HIPFFT_FORWARD);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftExecC2C(plan_batch)");

  nthreads = 1024;
  nblocks = npt / nthreads;
  if (npt % nthreads)
    nblocks++;

  k_multiply<<<nblocks,nthreads,0,stream>>> (buffer, d_kernel, niter);

  result = hipfftExecC2C (plan_bwd, buffer, buffer, HIPFFT_BACKWARD);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "Speed::runTest", "hipfftExecC2C(plan_callback)");

  hipfftComplex * ou = output;
  hipfftComplex * in = buffer;

  for (unsigned i=0; i<niter; i++)
  { 
    hipMemcpyAsync ((void *) ou, (void *) in, npt * sizeof(hipfftComplex), hipMemcpyDeviceToDevice, stream);
    ou += npt;
    in += npt;
  }

  hipStreamSynchronize(stream);

  timer_batch.stop ();

  total_time = timer_batch.get_elapsed();
  time_per_fft = total_time / niter;
  time_us = time_per_fft * 1e6;
  cerr << "BATCH: total_time=" << total_time << " time_per_fft=" << time_per_fft 
       << " time_us=" << time_us << endl;

  hipfftDestroy(plan_callback);
  hipfftDestroy(plan_batch);

  hipFree(raw);
  hipFree(input);
  hipFree(output);
  hipFree(d_kernel);
}
