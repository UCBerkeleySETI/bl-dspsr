#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2015 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/ConvolutionCUDASpectral.h"
#include "CUFFTError.h"
#include "debug.h"

#if HAVE_CUFFT_CALLBACKS
#include "dsp/ConvolutionCUDACallbacks.h"
#include <hipfft/hipfftXt.h>
#endif

#include <iostream>
#include <cassert>

using namespace std;

void check_error_stream (const char*, hipStream_t);

// ichan   == blockIdx.y
// ipt_bwd == blockIdx.x * blockDim.x + threadIdx.x
__global__ void k_multiply_conv_spectral (float2* d_fft, const __restrict__ float2 * kernel, unsigned npt_bwd)
{
  const unsigned idx = (blockIdx.y * npt_bwd) + (blockIdx.x * blockDim.x) + threadIdx.x;
  d_fft[idx] = hipCmulf(d_fft[idx], kernel[idx]);
}

// ichan == blockIdx.y
// ipt_bwd == blockIdx.x * blockDim.x + threadIdx.x
__global__ void k_ncopy_conv_spectral (float2* output_data, uint64_t ostride,
           const float2* input_data, uint64_t istride,
           unsigned nfilt_pos, unsigned nsamp_step)
{

  const unsigned idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (idx < nfilt_pos)
    return;

  uint64_t in_offset  = istride * blockIdx.y;
  uint64_t out_offset = ostride * blockIdx.y;

  unsigned isamp = idx;
  unsigned osamp = idx - nfilt_pos;

  if (osamp < nsamp_step)
    output_data[out_offset + osamp] = input_data[in_offset + isamp];
}

CUDA::ConvolutionEngineSpectral::ConvolutionEngineSpectral (hipStream_t _stream)
{
  stream = _stream;

  // create plan handles
  hipfftResult result;

  result = hipfftCreate (&plan_fwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::ConvolutionEngineSpectral", 
                      "hipfftCreate(plan_fwd)");

  result = hipfftCreate (&plan_bwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::ConvolutionEngineSpectral", 
                      "hipfftCreate(plan_bwd)");

  fft_configured = false;
  nchan = 0;
  npt_fwd = 0;
  npt_bwd = 0;

  work_area = 0;
  work_area_size = 0;

  buf = 0;
  d_kernels = 0;
}

CUDA::ConvolutionEngineSpectral::~ConvolutionEngineSpectral()
{
  hipfftResult result;

  result = hipfftDestroy (plan_fwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::~ConvolutionEngineSpectral",
                      "hipfftDestroy(plan_fwd)");

  result = hipfftDestroy (plan_bwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::~ConvolutionEngineSpectral",
                      "hipfftDestroy(plan_bwd)");

  if (work_area)
  {
    hipError_t error = hipFree (work_area);
    if (error != hipSuccess)
       throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::~ConvolutionEngineSpectral",
                    "hipFree(%xu): %s", &work_area,
                     hipGetErrorString (error));
  }

  if (buf)
  {
    hipError_t error = hipFree (buf);
    if (error != hipSuccess)
       throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::~ConvolutionEngineSpectral",
                    "hipFree(%xu): %s", &buf,
                     hipGetErrorString (error));
  }
}

void CUDA::ConvolutionEngineSpectral::regenerate_plans()
{
  hipfftResult result;
  result = hipfftDestroy (plan_fwd);
  result = hipfftCreate (&plan_fwd);

  result = hipfftDestroy (plan_bwd);
  result = hipfftCreate (&plan_bwd);
}

void CUDA::ConvolutionEngineSpectral::set_scratch (void * scratch)
{
  d_scratch = (hipfftComplex *) scratch;
}

// prepare all relevant attributes for the engine
void CUDA::ConvolutionEngineSpectral::prepare (dsp::Convolution * convolution)
{
  const dsp::Response* response = convolution->get_response();

  nchan = response->get_nchan();
  npt_bwd = response->get_ndat();
  npt_fwd = convolution->get_minimum_samples();
  nsamp_overlap = convolution->get_minimum_samples_lost();
  nsamp_step = npt_fwd - nsamp_overlap;
  nfilt_pos = response->get_impulse_pos ();
  nfilt_neg = response->get_impulse_neg ();

  if (convolution->get_input()->get_state() == Signal::Nyquist)
    type_fwd = HIPFFT_R2C;
  else
    type_fwd = HIPFFT_C2C;

  // configure the dedispersion kernel
  setup_kernel (convolution->get_response());

  fft_configured = false;

  // initialize the kernel size configuration
  mp.init();
  mp.set_nelement (npt_bwd);
}

// setup the convolution kernel based on the reposnse
void CUDA::ConvolutionEngineSpectral::setup_kernel (const dsp::Response * response)
{
  unsigned nchan = response->get_nchan();
  unsigned ndat = response->get_ndat();
  unsigned ndim = response->get_ndim();

  assert (ndim == 2);
  assert (d_kernels == 0);

  if (dsp::Operation::verbose)
    cerr << "CUDA::ConvolutionEngineSpectral::setup_kernel response: "
         << "nchan=" << nchan << " ndat=" << ndat << " ndim=" << ndim << endl;

	// allocate memory for dedispersion kernel of all channels
	unsigned kernels_size = ndat * sizeof(hipfftComplex) * nchan;
  hipError_t error = hipMalloc ((void**)&d_kernels, kernels_size);
  if (error != hipSuccess)
  {
    throw Error (InvalidState, "CUDA::ConvolutionEngineSpectral::setup_kernel",
     "could not allocate device memory for dedispersion kernel");
  }

  // copy all kernels from host to device
  const float* kernel = response->get_datptr (0,0);

  if (dsp::Operation::verbose)
    cerr << "CUDA::ConvolutionEngineSpectral::setup_kernel hipMemcpy stream=" 
         << stream << " size=" << kernels_size << endl;
  if (stream)
    error = hipMemcpyAsync (d_kernels, kernel, kernels_size, hipMemcpyHostToDevice, stream);
  else
    error = hipMemcpy (d_kernels, kernel, kernels_size, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    throw Error (InvalidState, "CUDA::ConvolutionEngineSpectral::setup_kernel",
     "could not copy dedispersion kernel to device");
  }

#if HAVE_CUFFT_CALLBACKS
  error = hipHostMalloc ((void **) h_conv_params, sizeof(unsigned) * 2);
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::ConvolutionEngineSpectral::setup_kernel",
                 "could not allocate memory for h_conv_params");

  h_conv_params[0] = nfilt_pos;
  h_conv_params[1] = npt_bwd - nfilt_neg;
  setup_callbacks_conv_params_spectral (h_conv_params, sizeof (h_conv_params), stream);
#endif
}

// configure the batched FFT plans
void CUDA::ConvolutionEngineSpectral::setup_batched (const dsp::TimeSeries* input,
                                                     dsp::TimeSeries * output)
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::ConvolutionEngineSpectral::setup_batched npt_fwd=" << npt_fwd 
         << " npt_bwd=" << npt_bwd << endl;

  nchan = input->get_nchan();
  npol  = input->get_npol();
  unsigned ndim = input->get_ndim();

#ifdef _DEBUG
  cerr << "CUDA::ConvolutionEngineSpectral::setup_batched nchan=" << nchan 
       << " npol=" << npol << " ndat=" << input->get_ndat() << endl;
#endif

  input_stride = (input->get_datptr (1, 0) - input->get_datptr (0, 0)) / ndim;
  output_stride = (output->get_datptr (1, 0) - output->get_datptr (0, 0) ) / ndim;

  int rank = 1; 
  int inembed[1];
  int onembed[1];
  int istride, ostride, idist, odist;
  hipfftResult result;

  // now setup the forward batched plan
  size_t work_size_fwd, work_size_bwd;

  // complex layout plans for input
  inembed[0] = npt_fwd;
  onembed[0] = npt_bwd;

  istride = 1;
  ostride = 1;

  idist = (int) input_stride;
  odist = npt_bwd;

#ifdef _DEBUG
  cerr << "CUDA::ConvolutionEngineSpectral::setup_batched npt_fwd=" << npt_fwd 
       << " nbatch=" << nchan << endl;
  cerr << "CUDA::ConvolutionEngineSpectral::setup_batched input_stride=" 
       << input_stride << " output_stride=" << output_stride << endl;
#endif

  // setup forward fft
  result = hipfftMakePlanMany (plan_fwd, rank, &npt_fwd, 
                              inembed, istride, idist,
                              onembed, ostride, odist,
                              type_fwd, nchan, &work_size_fwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched", 
                      "hipfftMakePlanMany (plan_fwd)");

  result = hipfftSetStream (plan_fwd, stream);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched",
          "hipfftSetStream(plan_fwd)");

  // get a rough estimate on work buffer size
  work_size_fwd = 0;
  result = hipfftEstimateMany(rank, &npt_fwd, 
                             inembed, istride, idist, 
                             onembed, ostride, odist, 
                             type_fwd, nchan, &work_size_fwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched",
                      "hipfftEstimateMany(plan_fwd)");

  istride = 1;
  ostride = 1;

#ifdef HAVE_CUFFT_CALLBACKS
  inembed[0] = npt_bwd;
  onembed[0] = nsamp_step;

  idist = npt_bwd;
  odist = (int) output_stride;
#else
  inembed[0] = npt_bwd;
  onembed[0] = npt_bwd;

  idist = npt_bwd;
  odist = npt_bwd;
#endif

  // the backward FFT is a has a simple layout (npt_bwd)
  DEBUG("CUDA::ConvolutionEngineSpectral::setup_batched hipfftMakePlanMany (plan_bwd)");
  result = hipfftMakePlanMany (plan_bwd, rank, &npt_bwd, 
                              inembed, istride, idist,
                              onembed, ostride, odist,
                              HIPFFT_C2C, nchan, &work_size_bwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched", 
                      "hipfftMakePlanMany (plan_bwd)");

  result = hipfftSetStream (plan_bwd, stream);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched",
                      "hipfftSetStream(plan_bwd)");

  DEBUG("CUDA::ConvolutionEngineSpectral::setup_batched bwd FFT plan set");

  work_size_bwd = 0;
  result = hipfftEstimateMany(rank, &npt_bwd, 
                             inembed, istride, idist, 
                             onembed, ostride, odist, 
                             HIPFFT_C2C, nchan, &work_size_bwd);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched",
                      "hipfftEstimateMany(plan_fwd)");
  
/*
  work_area_size = (work_size_fwd > work_size_bwd) ? work_size_fwd : work_size_bwd;
  auto_allocate = work_area_size > 0;

  DEBUG("CUDA::ConvolutionEngineSpectral::setup_batched hipfftSetAutoAllocation(plan_fwd)");
  result = hipfftSetAutoAllocation(plan_fwd, auto_allocate);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched",
                      "hipfftSetAutoAllocation(plan_bwd, %d)", 
                      auto_allocate);

  DEBUG("CUDA::ConvolutionEngineSpectral::setup_batched hipfftSetAutoAllocation(plan_bwd)");
  result = hipfftSetAutoAllocation(plan_bwd, auto_allocate);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_batched",
                      "hipfftSetAutoAllocation(plan_bwd, %d)", auto_allocate);

*/
  // free the space allocated for buf in setup_singular
  hipError_t error;
  if (buf)
  {
    error = hipFree (buf);
    if (error != hipSuccess)
      throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::setup_batched",
                   "hipFree(%x): %s", &buf, hipGetErrorString (error));
  }

  size_t batched_buffer_size = npt_bwd * nchan * sizeof (hipfftComplex);
  error = hipMalloc ((void **) &buf, batched_buffer_size);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::setup_batched",
                 "hipMalloc(%x, %u): %s", &buf, batched_buffer_size,
                 hipGetErrorString (error));

	// allocate device memory for dedispsersion kernel (1 channel)
/*
  if (work_area_size > 0)
  {
    if (work_area)
    {
      error = hipFree (work_area);
      if (error != hipSuccess)
         throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::setup",
                     "hipFree(%xu): %s", &work_area,
                     hipGetErrorString (error));
    }
    DEBUG("CUDA::ConvolutionEngineSpectral::setup hipMalloc("<<work_area<<", "<<work_area_size<<")");
    error = hipMalloc (&work_area, work_area_size);  
    if (error != hipSuccess)
      throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::setup", 
                   "hipMalloc(%x, %u): %s", &work_area, work_area_size,
                   hipGetErrorString (error));
  }
  else
    work_area = 0;
*/
}

// Perform convolution choosing the optimal batched size or if ndat is not as
// was configured, then perform singular
void CUDA::ConvolutionEngineSpectral::perform (const dsp::TimeSeries* input, dsp::TimeSeries * output, unsigned npart)
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::ConvolutionEngineSpectral::perform (" << npart << ")" << endl;

  if (npart == 0)
    return;

  uint64_t curr_istride = (input->get_datptr (1, 0) - input->get_datptr (0, 0)) / input->get_ndim();
  uint64_t curr_ostride = (output->get_datptr (1, 0) - output->get_datptr (0, 0)) / output->get_ndim();

  if (dsp::Operation::verbose)
  {
    cerr << "CUDA::ConvolutionEngineSpectral::perform istride prev=" << input_stride << " curr=" << curr_istride << " ndim=" << input->get_ndim() << endl;
    cerr << "CUDA::ConvolutionEngineSpectral::perform ostride prev=" << output_stride << " curr=" << curr_ostride << " ndim=" <<
output->get_ndim() << endl;
  }

  if (curr_istride != input_stride || curr_ostride != output_stride)
  {
    if (dsp::Operation::verbose)
      cerr << "CUDA::ConvolutionEngineSpectral::perform reconfiguring FFT batch sizes" << endl;
    fft_configured = false;
  }

  if (!fft_configured)
  {
    regenerate_plans ();
    setup_batched (input, output);
#if HAVE_CUFFT_CALLBACKS
    cerr << "CUDA::ConvolutionEngineSpectral::perform setup_callbacks_ConvolutionCUDASpectral()" << endl;
    setup_callbacks_ConvolutionCUDASpectral (plan_fwd, plan_bwd, d_kernels, stream);
#endif
    fft_configured = true;
  }

  if (type_fwd == HIPFFT_C2C)
  {
    perform_complex (input, output, npart);
  }
  else
  {
    cerr << "CUDA::ConvolutionEngineSpectral::perform_real not implemented" << endl;
    //perform_real (input, output, npart);
  }
}

void CUDA::ConvolutionEngineSpectral::perform_complex (const dsp::TimeSeries* input, 
                                                       dsp::TimeSeries * output,
                                                       unsigned npart)
{
  const unsigned npol = input->get_npol();
  const unsigned nchan = input->get_nchan();
  const unsigned ndim = input->get_ndim();
  const uint64_t ipol_stride = input_stride / npol;
  const uint64_t opol_stride = output_stride / npol;

  hipfftComplex * in;
  hipfftComplex * out;
  hipfftResult result;

	if (dsp::Operation::verbose)
  	cerr << "CUDA::ConvolutionEngineSpectral::perform_complex npart=" << npart 
				 << " nsamp_step=" << nsamp_step << endl;

#if !HAVE_CUFFT_CALLBACKS
  dim3 blocks = dim3 (npt_bwd / mp.get_nthread(), nchan);
  unsigned nthreads = mp.get_nthread();

  if (npt_bwd <= nthreads) 
  {
    blocks.x = 1;
    nthreads = npt_bwd;
  }
  else
  {
    if (npt_bwd % nthreads)
      blocks.x++;
  }
#endif

  hipfftComplex * in_t  = (hipfftComplex *) input->get_datptr (0, 0);
  hipfftComplex * out_t = (hipfftComplex *) output->get_datptr (0, 0);

  if (dsp::Operation::verbose)
    cerr << "CUDA::ConvolutionEngineSpectral::perform_complex in=" << in_t << " out=" << out_t << endl;

  for (unsigned ipart=0; ipart<npart; ipart++)
  {
    in  = in_t;
    out = out_t;

    for (unsigned ipol=0; ipol<npol; ipol++)
    {
      // perform nchan batched forward FFTs for the current ipol and ipart
      result = hipfftExecC2C (plan_fwd, in, buf, HIPFFT_FORWARD);
      if (result != HIPFFT_SUCCESS)
        throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::perform_complex",
                          "hipfftExecC2C(plan_fwd)");

#if HAVE_CUFFT_CALLBACKS
      // perform the inverse batched FFT (out-of-place)
      result = hipfftExecC2C (plan_bwd, buf, out, HIPFFT_BACKWARD);
      if (result != HIPFFT_SUCCESS)
        throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::perform_complex",
                            "hipfftExecC2C(plan_bwd)");

#else
      // multiply by the dedispersion kernel
      k_multiply_conv_spectral<<<blocks, nthreads, 0, stream>>> (buf, d_kernels, npt_bwd);

      // perform the inverse batched FFT (in-place)
      result = hipfftExecC2C (plan_bwd, buf, buf, HIPFFT_BACKWARD);
      if (result != HIPFFT_SUCCESS)
        throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::perform_complex",
                          "hipfftExecC2C(plan_bwd)");

      // copy batches of output from input
      k_ncopy_conv_spectral<<<blocks, nthreads, 0, stream>>> (out, output_stride,
                                                              buf, npt_bwd,
                                                              nfilt_pos, nsamp_step);
#endif
      in  += ipol_stride;
      out += opol_stride;
    }

    in_t  += nsamp_step;
    out_t += nsamp_step;
  }

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error_stream( "CUDA::ConvolutionEngineSpectral::perform_complex", stream );
}

#if 0
void CUDA::ConvolutionEngineSpectral::perform_real(const dsp::TimeSeries* input,
                                           dsp::TimeSeries * output,
                                           unsigned npart)
{
  const unsigned npol = input->get_npol();
  const unsigned nchan = input->get_nchan();
  const unsigned ndim = input->get_ndim();

  hipfftReal * in;
  hipfftComplex * out;
  hipfftResult result;

  const unsigned out_nsamp_step = nsamp_step / 2;

  const unsigned in_step_batch  = nsamp_step * nbatch;
  const unsigned out_step_batch = out_nsamp_step * nbatch;

  unsigned nbp = 0;
  if (nbatch > 0)
    nbp = npart / nbatch;

  dim3 blocks = dim3 (out_nsamp_step, nbatch, 0);
  if (out_nsamp_step % mp.get_nthread())
    blocks.x++;

  if (dsp::Operation::verbose)
    cerr << "CUDA::ConvolutionEngineSpectral::perform_real nsamp_step=" << nsamp_step
         << " npt_bwd=" << npt_bwd << endl;

  for (unsigned ichan=0; ichan<nchan; ichan++)
  {
    const unsigned k_offset = ichan * npt_bwd;

    for (unsigned ipol=0; ipol<npol; ipol++)
    {
      in  = (hipfftReal *) input->get_datptr (ichan, ipol);
      out = (hipfftComplex *) output->get_datptr (ichan, ipol);

      // for each batched FFT
      for (unsigned i=0; i<nbp; i++)
      {
        // perform forward batched FFT
        result = hipfftExecR2C (plan_fwd, in, buf);
        if (result != HIPFFT_SUCCESS)
          throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::perform_real",
                            "hipfftExecC2C(plan_fwd)");

        // multiply by the dedispersion kernel
        k_multiply_conv<<<mp.get_nblock(),mp.get_nthread(),0,stream>>> (buf,
                                                                   d_kernels + k_offset,
                                                                   nbatch);

        // perform the inverse batched FFT (in-place)
        result = hipfftExecC2C (plan_bwd, buf, buf, HIPFFT_BACKWARD);
        if (result != HIPFFT_SUCCESS)
          throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::perform_real",
                            "hipfftExecC2C(plan_bwd)");

        // copy batches of output from input
        k_ncopy_conv<<<blocks,mp.get_nthread(),0,stream>>> (out, out_nsamp_step,
                                                       buf + nfilt_pos, npt_bwd,
                                                       out_step_batch);

        in  += in_step_batch;
        out += out_step_batch;
      }

      for (unsigned ipart=nbp*nbatch; ipart<npart; ipart++)
      {
        result = hipfftExecR2C (plan_fwd, in, buf);
        if (result != HIPFFT_SUCCESS)
          throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::perform_real",
                            "hipfftExecC2C(plan_fwd)");

        // multiply by the dedispersion kernel
        k_multiply_conv<<<mp.get_nblock(),mp.get_nthread(),0,stream>>> (buf,
                                                                   d_kernels + k_offset,
                                                                   1);

        // perform the inverse batched FFT (in-place)
        result = hipfftExecC2C (plan_bwd, buf, buf, HIPFFT_BACKWARD);
        if (result != HIPFFT_SUCCESS)
          throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::perform",
                            "hipfftExecC2C(plan_bwd)");

        // copy batches of output from input
        k_ncopy_conv<<<blocks.x,mp.get_nthread(),0,stream>>> (out, out_nsamp_step,
                                                         buf + nfilt_pos, npt_bwd,
                                                         out_step_batch);
        in  += nsamp_step;
        out += out_nsamp_step;
      }
    }
  }
  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error_stream( "CUDA::ConvolutionEngineSpectral::perform_real", stream );
}
#endif
