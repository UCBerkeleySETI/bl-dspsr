//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2015 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/ConvolutionCUDACallbacks.h"
#include "CUFFTError.h"
#include "debug.h"

#if HAVE_CUFFT_CALLBACKS
#include <hipfft/hipfftXt.h>
#endif

using namespace std;

#if HAVE_CUFFT_CALLBACKS

// [0] channel offset ( ichan * npt)
// [1] npt
// [2] first_ipt ( nfilt_pos )
// [3] last_ipt ( npt - nfilt_neg )
// [4] nfilt_tot 
__device__ __constant__ unsigned conv_params[5];

/////////////////////////////////////////////////////////////////////////
//
// store with multiplication by dedispersion kernel [no FFT batching]
//
__device__ void CB_convolve_and_store (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  // the dedispersion kernel complex float for this element of the FFT
  const hipfftComplex k = ((hipfftComplex *) callerInfo)[conv_params[0] + offset];
  ((hipfftComplex*)dataOut)[offset] = hipCmulf (d, k);
}

__device__ void CB_convolve_and_store_batch (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  // the dedispersion kernel value for this element of the FFT
  const unsigned kernel_offset = conv_params[0] + (offset % conv_params[1]);
  const hipfftComplex k = ((hipfftComplex *) callerInfo)[kernel_offset];

  ((hipfftComplex*)dataOut)[offset] = hipCmulf (d, k);
}
__device__ hipfftCallbackStoreC d_store_fwd        = CB_convolve_and_store;
__device__ hipfftCallbackStoreC d_store_fwd_batch = CB_convolve_and_store_batch;

/////////////////////////////////////////////////////////////////////////
//
// store with output filtering on
//
__device__ void CB_filtered_store (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  // if offset < nfilt_pos, discard
  if (offset < conv_params[2])
    return;

  // if offset > (npt - nfilt_neg), discard
  if (offset >= conv_params[3])
    return;

  ((hipfftComplex*)dataOut)[offset - conv_params[2]] = d;
}

__device__ void CB_filtered_store_batch (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  const unsigned ibatch = offset / conv_params[1];
  const unsigned ipt = offset - (ibatch * conv_params[1]);

  // if ipt < nfilt_pos, discard
  if (ipt < conv_params[2])
    return;
  
  // if ipt > (npt - nfilt_neg), discard
  if (ipt >= conv_params[3])
    return;

  // substract the required offsets
  offset -= ((ibatch * conv_params[4]) + conv_params[2]);

  ((hipfftComplex*)dataOut)[offset] = d;
}

__device__ hipfftCallbackStoreC d_store_bwd       = CB_filtered_store;
__device__ hipfftCallbackStoreC d_store_bwd_batch = CB_filtered_store_batch;

void setup_callbacks_ConvolutionCUDA (hipfftHandle plan_fwd, hipfftHandle plan_bwd, 
                                      hipfftHandle plan_fwd_batched, hipfftHandle plan_bwd_batched,
                                      hipfftComplex * d_kernels, int nbatch, hipStream_t stream)
{
  hipError_t error;
  hipfftResult_t result;

  hipfftCallbackStoreC h_store_fwd;
  hipfftCallbackStoreC h_store_bwd;
  hipfftCallbackStoreC h_store_fwd_batch;
  hipfftCallbackStoreC h_store_bwd_batch;

  error = hipMemcpyFromSymbolAsync(&h_store_fwd, HIP_SYMBOL(d_store_fwd), 
                                    sizeof(h_store_fwd), 0, 
                                    hipMemcpyDeviceToHost, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::ConvolutionEngine::setup_callbacks",
                 "hipMemcpyFromSymbolAsync failed for h_store_fwd");

  error = hipMemcpyFromSymbolAsync(&h_store_bwd, HIP_SYMBOL(d_store_bwd),
                                    sizeof(h_store_bwd), 0,
                                    hipMemcpyDeviceToHost, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::ConvolutionEngine::setup_callbacks",
                 "hipMemcpyFromSymbolAsync failed for h_store_bwd");

  error = hipMemcpyFromSymbolAsync(&h_store_fwd_batch, HIP_SYMBOL(d_store_fwd_batch),
                                    sizeof(h_store_fwd_batch), 0,
                                    hipMemcpyDeviceToHost, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::ConvolutionEngine::setup_callbacks",
                 "hipMemcpyFromSymbolAsync failed for h_store_fwd_batch");

  error = hipMemcpyFromSymbolAsync(&h_store_bwd_batch, HIP_SYMBOL(d_store_bwd_batch),
                                    sizeof(h_store_bwd_batch), 0,
                                    hipMemcpyDeviceToHost, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::ConvolutionEngine::setup_callbacks",
                 "hipMemcpyFromSymbolAsync failed for h_store_bwd_batch");

  result = hipfftXtSetCallback (plan_fwd, (void **)&h_store_fwd,
                               HIPFFT_CB_ST_COMPLEX, (void **)&d_kernels);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
      "hipfftXtSetCallback (plan_fwd, h_store_fwd)");

  result = hipfftXtSetCallback (plan_bwd, (void **)&h_store_bwd,
                               HIPFFT_CB_ST_COMPLEX, 0);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
      "hipfftXtSetCallback (plan_bwd, h_store_bwd)");

  if (nbatch > 0)
  {
    result = hipfftXtSetCallback (plan_fwd_batched, (void **)&h_store_fwd_batch,
                                 HIPFFT_CB_ST_COMPLEX, (void **)&d_kernels);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
        "hipfftXtSetCallback (plan_fwd_batched, h_store_fwd_batch)");

    result = hipfftXtSetCallback (plan_bwd_batched, (void **)&h_store_bwd_batch,
                                 HIPFFT_CB_ST_COMPLEX, 0);
    if (result != HIPFFT_SUCCESS)
      throw CUFFTError (result, "CUDA::ConvolutionEngine::setup_callbacks",
        "hipfftXtSetCallback (plan_bwd_batched, h_store_bwd_batch)");
  }
}

void setup_callbacks_conv_params (unsigned * h_ptr, unsigned h_size, hipStream_t stream)
{
  hipError_t error = hipMemcpyToSymbolAsync(HIP_SYMBOL(conv_params), (void *) h_ptr,
                                   h_size, 0,
                                   hipMemcpyHostToDevice, stream);
  if (error != hipSuccess)
  {
    throw Error (InvalidState, "CUDA::ConvolutionEngine::setup_kernel",
     "could not initialize convolution params in device memory");
  }

}

// 
//
//

// [0] first_ipt ( nfilt_pos )
// [1] last_ipt ( npt - nfilt_neg )
__device__ __constant__ unsigned conv_params_spectral[2];

/////////////////////////////////////////////////////////////////////////
//
// store with multiplication by dedispersion kernel
//
__device__ void CB_convolve_and_store_spectral (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  // the dedispersion kernel complex float for this element of the FFT
  const hipfftComplex k = ((hipfftComplex *) callerInfo)[offset];
  ((hipfftComplex*)dataOut)[offset] = hipCmulf (d, k);
}
__device__ hipfftCallbackStoreC d_store_fwd_spectral = CB_convolve_and_store_spectral;

/////////////////////////////////////////////////////////////////////////
//
// store with output filtering on
//
__device__ void CB_filtered_store_spectral (void * dataOut, size_t offset, hipfftComplex d, void * callerInfo, void *sharedPtr)
{
  // if offset < nfilt_pos, discard
  if (offset < conv_params_spectral[0])
    return;

  // if offset > (npt - nfilt_neg), discard
  if (offset >= conv_params_spectral[1])
    return;

  ((hipfftComplex*)dataOut)[offset - conv_params_spectral[0]] = d;
}
__device__ hipfftCallbackStoreC d_store_bwd_spectral = CB_filtered_store_spectral;


void setup_callbacks_ConvolutionCUDASpectral (hipfftHandle plan_fwd, hipfftHandle plan_bwd, hipfftComplex * d_kernels, hipStream_t stream)
{
  hipError_t error;
  hipfftResult_t result;

  hipfftCallbackStoreC h_store_fwd;
  hipfftCallbackStoreC h_store_bwd;

  error = hipMemcpyFromSymbolAsync(&h_store_fwd, HIP_SYMBOL(d_store_fwd_spectral),
                                    sizeof(h_store_fwd), 0,
                                    hipMemcpyDeviceToHost, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::setup_callbacks",
                 "hipMemcpyFromSymbolAsync failed for h_store_fwd");

  error = hipMemcpyFromSymbolAsync(&h_store_bwd, HIP_SYMBOL(d_store_bwd_spectral),
                                    sizeof(h_store_bwd), 0,
                                    hipMemcpyDeviceToHost, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::ConvolutionEngineSpectral::setup_callbacks",
                 "hipMemcpyFromSymbolAsync failed for h_store_bwd");

  result = hipfftXtSetCallback (plan_fwd, (void **)&h_store_fwd,
                               HIPFFT_CB_ST_COMPLEX, (void **)&d_kernels);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_callbacks",
      "hipfftXtSetCallback (plan_fwd, h_store_fwd)");

  result = hipfftXtSetCallback (plan_bwd, (void **)&h_store_bwd,
                               HIPFFT_CB_ST_COMPLEX, 0);
  if (result != HIPFFT_SUCCESS)
    throw CUFFTError (result, "CUDA::ConvolutionEngineSpectral::setup_callbacks",
      "hipfftXtSetCallback (plan_bwd, h_store_bwd)");
}

void setup_callbacks_conv_params_spectral (unsigned * h_ptr, unsigned h_size, hipStream_t stream)
{
  hipError_t error = hipMemcpyToSymbolAsync(HIP_SYMBOL(conv_params_spectral), (void *) h_ptr,
                                   h_size, 0, hipMemcpyHostToDevice, stream);
  if (error != hipSuccess)
  {
    throw Error (InvalidState, "CUDA::ConvolutionEngineSpectral::setup_kernel",
     "could not initialize convolution params in device memory");
  }
}




#endif
