#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2015 by Matthew Kerr
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/TScrunchCUDA.h"

#include <hip/hip_complex.h>
#include "Error.h"
#include "debug.h"

//#include <memory>

//#include <string.h>

using namespace std;

void check_error (const char*);

CUDA::TScrunchEngine::TScrunchEngine (hipStream_t _stream)
{
  stream = _stream;
}

__global__ void fpt_ndim1_ndim1 (float* in_base, float* out_base,
    unsigned in_Fstride, unsigned in_Pstride,
    unsigned out_Fstride, unsigned out_Pstride,
    unsigned output_ndat, unsigned sfactor)
{

  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= output_ndat)
    return;

  // blockIdx.y == channel index
  // threadIdx.y == polarization index
  // offset into buffer = the index of the output datum (i) * the scrunch factor
  in_base += blockIdx.y * in_Fstride + threadIdx.y * in_Pstride + sfactor * i;
  float result = *in_base;
  for (int j=1; j < sfactor; ++j,++in_base)
  {
    result += *in_base;
  }

  out_base += blockIdx.y * out_Fstride +threadIdx.y * out_Pstride + i;
  *out_base = result;
}

__global__ void fpt_ndim2_ndim2 (float2* in_base, float2* out_base,
    unsigned in_Fstride, unsigned in_Pstride, 
    unsigned out_Fstride, unsigned out_Pstride,
    unsigned output_ndat, unsigned sfactor)
{

  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= output_ndat)
    return;

  // blockIdx.y == channel index
  // threadIdx.y == polarization index
  // offset into buffer = the index of the output datum (i) * the scrunch factor
  in_base += blockIdx.y * in_Fstride + threadIdx.y * in_Pstride + sfactor * i;
  float2 result = *in_base;
  for (int j=1; j < sfactor; ++j,++in_base)
  {
    result.x += (*in_base).x;
    result.y += (*in_base).y;
  }

  out_base += blockIdx.y * out_Fstride +threadIdx.y * out_Pstride + i;
  *out_base = result;
}

__global__ void fpt_ndim1_ndim1_shm (float* in_base, float* out_base,
    unsigned in_Fstride, unsigned in_Pstride,
    unsigned out_Fstride, unsigned out_Pstride,
    unsigned ndat_out, unsigned sfactor)
{
  // shared memory for coalesced reads
  extern __shared__ float ndim1_shm[];

  // blockIdx.y == channel index
  // threadIdx.y == polarization index
  unsigned ndat_in = ndat_out * sfactor;

  const unsigned block_offset = blockIdx.x * blockDim.x * sfactor;

  // X dimension is indexed on output samples. This is the input sample each thread will start to read
  unsigned isamp_thr = block_offset + threadIdx.x;

  // offset into buffer = the index the first read sample for this block
  in_base += (blockIdx.y*in_Fstride) + (threadIdx.y*in_Pstride) + block_offset;

  float result = 0;
  unsigned isamp = threadIdx.x * sfactor;
  unsigned esamp = isamp + sfactor;
  unsigned shm_start = 0;
  unsigned shm_end = blockDim.x;

  // ensure we don't overshoot the number of ndat
  for (unsigned j=0; j<sfactor; j++)
  {
    // just whole block to coalesce read into SHM
    if (isamp_thr < ndat_in)
      ndim1_shm[threadIdx.x] = in_base[isamp_thr];

    __syncthreads();

    // each thread adds time samples into its output result, wait for
    // the right time samples to be located in shm

    // if this thread's output value is located in SHM, add to result
    while (isamp >= shm_start && isamp < shm_end && isamp < esamp)
    {
      //if (blockIdx.y == 0 && blockIdx.z == 0) 
      //  printf ("[%d][%d] isamp=%u esamp=%u start=%u end=%u\n", blockIdx.x, threadIdx.x, isamp, esamp, shm_start, shm_end);
      result = result + ndim1_shm[isamp-shm_start];
      isamp++;
    }

    isamp_thr += blockDim.x;
    shm_start += blockDim.x;
    shm_end   += blockDim.x;
  }

  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= ndat_out)
    return;

  //if (blockIdx.y == 0 && blockIdx.z == 0)
  //  printf ("[%d][%d] i=%u\n", blockIdx.x, threadIdx.x, i);

  out_base += (blockIdx.y*out_Fstride) + (threadIdx.y*out_Pstride) + i;
  *out_base = result;
}


__global__ void fpt_ndim2_ndim2_shm (float2* in_base, float2* out_base,
    unsigned in_Fstride, unsigned in_Pstride,
    unsigned out_Fstride, unsigned out_Pstride,
    unsigned ndat_out, unsigned sfactor)
{
  // shared memory for coalesced reads
  extern __shared__ hipFloatComplex ndim2_shm[];

  // blockIdx.y == channel index
  // threadIdx.y == polarization index
  unsigned ndat_in = ndat_out * sfactor;

  const unsigned block_offset = blockIdx.x * blockDim.x * sfactor;

  // X dimension is indexed on output samples. This is the input sample each thread will start to read
  unsigned isamp_thr = block_offset + threadIdx.x;
 
  // offset into buffer = the index the first read sample for this block
  in_base += (blockIdx.y*in_Fstride) + (threadIdx.y*in_Pstride) + block_offset;

  hipFloatComplex result = make_hipComplex(0,0);
  unsigned isamp = threadIdx.x * sfactor;
  unsigned esamp = isamp + sfactor;
  unsigned shm_start = 0;
  unsigned shm_end = blockDim.x;

  // ensure we don't overshoot the number of ndat
  for (unsigned j=0; j<sfactor; j++)
  {
    // just whole block to coalesce read into SHM
    if (isamp_thr < ndat_in)
      ndim2_shm[threadIdx.x] = in_base[isamp_thr];

    __syncthreads();

    // each thread adds time samples into its output result, wait for
    // the right time samples to be located in shm

    // if this thread's output value is located in SHM, add to result
    while (isamp >= shm_start && isamp < shm_end && isamp < esamp)
    {
      //if (blockIdx.y == 0 && blockIdx.z == 0) 
      //  printf ("[%d][%d] isamp=%u esamp=%u start=%u end=%u\n", blockIdx.x, threadIdx.x, isamp, esamp, shm_start, shm_end);
      result = hipCaddf (result, ndim2_shm[isamp-shm_start]);
      isamp++;
    }

    isamp_thr += blockDim.x;
    shm_start += blockDim.x;
    shm_end   += blockDim.x;
  }
 
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= ndat_out)
    return;

  //if (blockIdx.y == 0 && blockIdx.z == 0)
  //  printf ("[%d][%d] i=%u\n", blockIdx.x, threadIdx.x, i);

  out_base += (blockIdx.y*out_Fstride) + (threadIdx.y*out_Pstride) + i;
  *out_base = result;
}


void CUDA::TScrunchEngine::fpt_tscrunch(const dsp::TimeSeries *in,
    dsp::TimeSeries* out, unsigned sfactor)
{
  // the "inner loop", which each thread does, is the tscrunch itself
  
  // the theory is that if one is time scrunching on the GPU, the 
  // scrunch factor will be something of order 10-100, a reasonable
  // amount of work for a thread to do

  // this is not at all optimal in terms of cache access, and at some point
  // this should be re-written with each thread accessing adjacent samples

  // to manage restrictions on grid size in earlier compute capability,
  // use a 2d thread block with one dimension corresponding to P,
  // the other to the output T
  // then launch on a 2d grid with one block handling the full output size, the
  // other handling the channels

  // each thread is assigned to a specific input F & input P
  // and loops over input T to add sfactor data together 

  // currently the only implementation uses float2s so we require
  // ndim==2 for both input and output
  //if (in->get_ndim() != 2)
  //  throw Error (InvalidParam, "CUDA::TScrunchEngine::fpt_scrunch",
  //   "cannot handle ndim=%u != 2", in->get_ndim());

  //if (out->get_ndim() != 2)
  //  throw Error (InvalidParam, "CUDA::TScrunchEngine::fpt_scrunch",
  //		 "cannot handle ndim=%u != 2", in->get_ndim());

  if (in->get_ndim() != out->get_ndim())
  {
    throw Error (InvalidParam, "CUDA::TScrunchEngine::fpt_scrunch",
       "cannot handle input ndim=%u != output ndim=%u", in->get_ndim()),
       out->get_ndim();
  }

  if (out == in)
    throw Error (InvalidParam, "CUDA::TScrunchEngine::fpt_scrunch",
		 "only out-of-place transformation implemented");

  if (in->get_ndat() == 0)
    return;

  unsigned ndim = in->get_ndim();

  uint64_t in_Fstride = (in->get_datptr(1)-in->get_datptr(0)) / ndim;
  uint64_t out_Fstride = (out->get_datptr(1)-out->get_datptr(0)) / ndim;

  uint64_t in_Pstride = 0;
  uint64_t out_Pstride = 0;
  if (in->get_npol() > 1)
  {
    in_Pstride = (in->get_datptr(0,1)-in->get_datptr(0,0)) / ndim;
    out_Pstride = (out->get_datptr(0,1)-out->get_datptr(0,0)) / ndim;
  }

  // use a 2-dimensional thread block to eliminate 3rd grid dimension

#define USE_SHARED
#ifdef USE_SHARED
  // set number of threads to be number of output samples, cap at 512
  dim3 threads (512);
  if (out->get_ndat() < 512)
    threads.x = out->get_ndat();
  dim3 blocks (out->get_ndat()/threads.x, in->get_nchan(), in->get_npol());
  if (out->get_ndat() % threads.x)
    blocks.x ++;

  if (ndim == 2)
  {
    size_t shm_bytes = threads.x * sizeof(float2);
    fpt_ndim2_ndim2_shm<<<blocks,threads,shm_bytes,stream>>> (
      (float2*)(in->get_datptr(0)), (float2*)(out->get_datptr(0)), 
      in_Fstride, in_Pstride, out_Fstride, out_Pstride, 
      out->get_ndat(), sfactor);
  }
  else
  {
    size_t shm_bytes = threads.x * sizeof(float);
    fpt_ndim1_ndim1_shm<<<blocks,threads,shm_bytes,stream>>> (
      (float*)(in->get_datptr(0)), (float*)(out->get_datptr(0)),
      in_Fstride, in_Pstride, out_Fstride, out_Pstride,
      out->get_ndat(), sfactor);
  }
#else
  dim3 threads (128, in->get_npol());
  dim3 blocks (out->get_ndat()/threads.x, in->get_nchan(), in->get_npol());
  if (out->get_ndat() % threads.x)
    blocks.x ++;
  if (ndim == 2)
  {
    fpt_ndim2_ndim2<<<blocks,threads,0,stream>>> (
      (float2*)(in->get_datptr(0)), (float2*)(out->get_datptr(0)), 
      in_Fstride, in_Pstride, out_Fstride, out_Pstride, 
      out->get_ndat(), sfactor);
  }
  else
  {
    fpt_ndim1_ndim1<<<blocks,threads,0,stream>>> (
      (float*)(in->get_datptr(0)), (float*)(out->get_datptr(0)), 
      in_Fstride, in_Pstride, out_Fstride, out_Pstride, 
      out->get_ndat(), sfactor);
  }
#endif


  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error ("CUDA::TScrunchEngine::fpt_scrunch");
}

