#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2016 by Andre Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "dsp/SKComputerCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "Error.h"
#include "templates.h"
#include "debug.h"

#include <stdio.h>
#include <memory>
#include <string.h>

#include <hip/hip_complex.h>

#ifdef __CUDA_ARCH__
    #if (__CUDA_ARCH__ >= 300)
        #define HAVE_SHFL
    #else
        #define NO_SHFL
    #endif
#endif

using namespace std;

void check_error (const char*);
void check_error_stream (const char*, hipStream_t);

/*
 *  Important Note, this engine is only efficient for larger strides (256-512)
 *  stride == nbeam for molongolo
 */

CUDA::SKComputerEngine::SKComputerEngine (dsp::Memory * memory)
{
  device_memory = dynamic_cast<CUDA::DeviceMemory *>(memory);
  stream = device_memory->get_stream();

  work_buffer_size = 0;
  work_buffer = 0;
}

void CUDA::SKComputerEngine::setup ()
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::SKComputerEngine::setup ()" << endl;

  // determine GPU capabilities
  int device = 0;
  hipGetDevice(&device);
  struct hipDeviceProp_t device_properties;
  hipGetDeviceProperties (&device_properties, device);
  max_threads_per_block = device_properties.maxThreadsPerBlock;
}

// each
__global__ void reduce_sqld_new (float2 * in, float2 * sums, float * skestimates, uint64_t in_stride, unsigned M)
{
  extern __shared__ float s1s[];
  float * s2s = s1s + 32;

  // each block integrates M samples
  const unsigned ichanpol = blockIdx.y;
  const unsigned nchanpol = gridDim.y;

  // offset to current channel, pol
  in += (ichanpol * in_stride) + (blockIdx.x * M);

  float power;
  float2 val;
  float s1 = 0;
  float s2 = 0;

  // in case M is > blockDim.x
  for (unsigned i=threadIdx.x; i<M; i+=blockDim.x)
  {
    // load the complex value 
    val = in[i];

    power = (val.x * val.x) + (val.y * val.y);
    s1 += power;
    s2 += (power * power);
  }

#ifdef HAVE_SHFL
  s1 += __shfl_down (s1, 16);
  s1 += __shfl_down (s1, 8);
  s1 += __shfl_down (s1, 4);
  s1 += __shfl_down (s1, 2);
  s1 += __shfl_down (s1, 1);

  s2 += __shfl_down (s2, 16);
  s2 += __shfl_down (s2, 8);
  s2 += __shfl_down (s2, 4);
  s2 += __shfl_down (s2, 2);
  s2 += __shfl_down (s2, 1);

  unsigned warp_idx = threadIdx.x % 32;
  unsigned warp_num = threadIdx.x / 32;
  
  if (warp_idx == 0)
  {
    s1s[warp_num] = s1;
    s2s[warp_num] = s2;
  }
  __syncthreads(); 

  if (warp_num == 0)
  {
    s1 = s1s[warp_idx];
    s2 = s2s[warp_idx];

    s1 += __shfl_down (s1, 16);
    s1 += __shfl_down (s1, 8);
    s1 += __shfl_down (s1, 4);
    s1 += __shfl_down (s1, 2);
    s1 += __shfl_down (s1, 1);

    s2 += __shfl_down (s2, 16);
    s2 += __shfl_down (s2, 8);
    s2 += __shfl_down (s2, 4);
    s2 += __shfl_down (s2, 2);
    s2 += __shfl_down (s2, 1);

    // s1 and s2 sums across block are complete
    if (warp_idx == 0)
    {
      val.x = s1;
      val.y = s2;
      unsigned odx = blockIdx.x*nchanpol + ichanpol;
      sums [odx] = val;
      skestimates[odx] = ((M+1) / (M-1)) * (M * (s2 / (s1 * s1)) - 1);
    }
  }
#endif
#ifdef NO_SHFL

  s1s[threadIdx.x] = s1;
  s2s[threadIdx.x] = s2;

  __syncthreads();

  int last_offset = blockDim.x/2;
  for (int offset = last_offset; offset > 0;  offset >>= 1)
  {
    if (threadIdx.x < offset)
    {
      s1s[threadIdx.x] += s1s[threadIdx.x + offset];
      s2s[threadIdx.x] += s2s[threadIdx.x + offset];
    }
    __syncthreads();
  }

  if (threadIdx.x == 0)
  {
    val.x = s1s[0];
    val.y = s2s[0];
    unsigned odx = blockIdx.x*nchanpol + ichanpol;
    sums [odx] = val;
    skestimates[odx] = ((M+1) / (M-1)) * (M * (val.y / (val.x * val.x)) - 1);
  }


#endif

  // now we need to a reduction across the block
}


/* Perform a reduction including SQLD calculations */
__global__ void reduce_sqld (float * in, float * out, const uint64_t ndat)
{
  extern __shared__ float sdata[];

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int s1 = (threadIdx.x*2);
  unsigned int s2 = (threadIdx.x*2) + 1;

  float re = 0;
  float im = 0;
  if (i < ndat)
  {
    re = in[(2*i)];
    im = in[(2*i) + 1];
  }

  sdata[s1] = (re * re) + (im * im);
  sdata[s2] = sdata[s1] * sdata[s1];

  __syncthreads();

  int last_offset = blockDim.x/2 + blockDim.x % 2;

  for (int offset = blockDim.x/2; offset > 0;  offset >>= 1)
  {
    // add a partial sum upstream to our own
    if (threadIdx.x < offset)
    {
      sdata[s1] += sdata[s1 + (2*offset)];
      sdata[s2] += sdata[s2 + (2*offset)];
    }
    __syncthreads();

    // special case for non power of 2 reductions
    if ((last_offset % 2) && (last_offset > 2) && (threadIdx.x == offset))
    {
      sdata[0] += sdata[s1 + (2*offset)];
      sdata[1] += sdata[s2 + (2*offset)];
    }

    last_offset = offset;

    // wait until all threads in the block have updated their partial sums
    __syncthreads();
  }

  // thread 0 writes the final result
  if (threadIdx.x == 0)
  {
    out[(2*blockIdx.x)]   = sdata[0];
    out[(2*blockIdx.x)+1] = sdata[1];
  }
}

/* sum each set of S1 and S2 and compute SK estimate for whole block */
__global__ void reduce_sk_estimate_new (float2* input, float * output, unsigned nchanpol, unsigned ndat, float M)
{
  // input are stored in TFP order
  const float M_fac = (M+1) / (M-1);

  for (unsigned ichanpol=threadIdx.x; ichanpol<nchanpol; ichanpol+=blockDim.x)
  {
    float2* in = input;
    float2 sum = make_hipComplex(0,0);;

    for (unsigned idat=0; idat<ndat; idat++)
    {
      sum = hipCaddf (sum, in[ichanpol]);
      in += nchanpol;
    }
    output[ichanpol] = M_fac * (M * (sum.y/ (sum.x * sum.x)) - 1);
  }
}


__global__ void reduce_sk_estimate (float * in, float * out, const uint64_t ndat, float M, unsigned ichan)
{
  extern __shared__ float sdata[];

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int s1 = (threadIdx.x*2);
  unsigned int s2 = (threadIdx.x*2) + 1;

  // load input into shared memory
  float re = 0;
  float im = 0;
  if (i < ndat)
  {
    re = in[(2*i)];
    im = in[(2*i) + 1];
  }

  sdata[s1] = re;
  sdata[s2] = im;

  __syncthreads();

  int last_offset = blockDim.x/2 + blockDim.x % 2;
  for (int offset = blockDim.x/2; offset > 0;  offset >>= 1)
  {
    // add a partial sum upstream to our own
    if (threadIdx.x < offset)
    {
      sdata[s1] += sdata[s1 + (2*offset)];
      sdata[s2] += sdata[s2 + (2*offset)];
    }

    __syncthreads();

    // special case for non power of 2 reductions
    if ((last_offset % 2) && (last_offset > 2) && (threadIdx.x == offset))
    {
      sdata[0] += sdata[s1 + (2*offset)];
      sdata[1] += sdata[s2 + (2*offset)];
    }

    last_offset = offset;

    // wait until all threads in the block have updated their partial sums
    __syncthreads();
  }

  // thread 0 writes the final result
  if (threadIdx.x == 0)
  {
    if (sdata[0] == 0)
      out[0] = 0;
    else
    {
      float M_fac = (M+1) / (M-1);
      out[0] = M_fac * (M * (sdata[1] / (sdata[0]*sdata[0])) - 1);
    }
  }
}

__global__ void calc_sk_estimate (float * in, float * out, float M_fac, unsigned int M, size_t out_span)
{
  unsigned int i = threadIdx.x;
  float S1_sum = in[(2*i)];
  float S2_sum = in[(2*i)+1];
  if (S1_sum == 0)
    out[out_span*i] = 0;
  else
    out[out_span*i] = M_fac * (M * (S2_sum / (S1_sum * S1_sum)) - 1);
}

// calculate SK statistics
void CUDA::SKComputerEngine::compute (const dsp::TimeSeries* input,
           dsp::TimeSeries* output, dsp::TimeSeries *output_tscr, unsigned M)
{
  if (dsp::Operation::verbose)
    std::cerr << "CUDA::SKComputerEngine::compute()" << std::endl;

  const uint64_t ndat = output->get_ndat() * M;
  const unsigned nchan = input->get_nchan ();
  const unsigned npol  = input->get_npol ();
  const unsigned nchanpol = nchan * npol;

  if (dsp::Operation::verbose)
    std::cerr << "CUDA::SKComputerEngine::compute ndat=" << ndat << " nchan="
              << nchan << " npol=" << npol << " M=" << M << std::endl;

  float * outdat = output->get_dattfp();
  float * outdat_tscr = output_tscr->get_dattfp();
  if (dsp::Operation::verbose)
  {
    std::cerr << "CUDA::SKComputerEngine::compute outdat=" << (void *) outdat << endl;
    std::cerr << "CUDA::SKComputerEngine::compute outdat_tscr=" << (void *) outdat_tscr << endl;
  }

  // TODO: currently only support FPT on GPU due to FoldCUDA
  switch (input->get_order())
  {
    case dsp::TimeSeries::OrderFPT:
    {
      if (dsp::Operation::verbose)
        std::cerr << "CUDA::SKComputerEngine::compute OrderFPT" << std::endl;

      float2 * indat = (float2*) input->get_datptr (0, 0);

      unsigned nthreads = 1024;
      if (M < nthreads)
        nthreads = M;
      dim3 blocks (ndat / M, nchanpol);

      // this is by design, due to input buffering
      assert (ndat % M == 0);

      // work buffer for S1 and S2 values for each set of M samples 
      size_t bytes_required = nchanpol * blocks.x * sizeof(float2);
      if (bytes_required > work_buffer_size)
      {
        if (work_buffer)
        {
          hipFree(work_buffer);
        }
        work_buffer_size = bytes_required;
        hipMalloc (&work_buffer, work_buffer_size);
      }

      if (dsp::Operation::verbose)
        cerr << "CUDA::SKComputerEngine::compute ndat=" << ndat
             << " blocks=(" << blocks.x << "," << blocks.y << ")"
             << " nthreads=" << nthreads << endl; 

      // require an S1 and S2 value for each warp in each block
      size_t shm_bytes_1 = 32 * sizeof(float2);

      if (dsp::Operation::verbose)
        cerr << "CUDA::SKComputerEngine::compute work_buffer=" << (void *) work_buffer << endl;

      uint64_t in_stride;
      if (npol > 1)
        in_stride = input->get_datptr (0, 1) - input->get_datptr (0, 0);
      else
        in_stride = input->get_datptr (1, 0) - input->get_datptr (0, 0);

      // for float2
      in_stride /= 2;

      reduce_sqld_new<<<blocks,nthreads,shm_bytes_1,stream>>> ((float2 *) indat, (float2 *) work_buffer, outdat, in_stride, M);
      if (dsp::Operation::record_time || dsp::Operation::verbose)
        if (stream)
          check_error_stream ("CUDA::SKComputerEngine::compute reduce_sqld_new [first]", stream);
      else
        check_error ("CUDA::SKComputerEngine::compute reduce_sqld_new [first]");

      // compute a tscrunched output SK
      nthreads = 1024;
      if (nchanpol < nthreads)
        nthreads = nchanpol;
      reduce_sk_estimate_new<<<1,nthreads,0,stream>>>((float2*) work_buffer, outdat_tscr, nchanpol, blocks.x, ndat);

#if 0


      // TODO consider making ichan a ydim?
      for (unsigned ichan=0; ichan<nchan; ichan++)
      {
        for (unsigned ipol=0; ipol<npol; ipol++)
        {
          indat = const_cast<float*>(input->get_datptr (ichan, ipol));

          //cerr << "CUDA::SKComputerEngine::compute ichan=" << ichan << " pol=" << ipol << " indat=" << indat << endl;

          // foreach block reduce to S1, S2 sums [out of place]
          //cerr << "CUDA::SKComputerEngine::compute [1] [" << ichan << ", " << ipol << "] shm_bytes=" << shm_bytes_1 << endl;
          reduce_sqld<<<nblocks,block_size,shm_bytes_1, stream>>> (indat, work_buffer, ndat_proc);
          if (dsp::Operation::record_time || dsp::Operation::verbose)
            if (stream)
              check_error_stream ("CUDA::SKComputerEngine::compute reduce_sqld [first]", stream);
            else
              check_error ("CUDA::SKComputerEngine::compute reduce_sqld [first]");

          // calculate S1, S2 sums for tscr [in place]
          //cerr << "CUDA::SKComputerEngine::compute [2] [" << ichan << ", " << ipol << "] shm_bytes=" << shm_bytes_2 << endl;
          reduce_sk_estimate<<<1,nblocks,shm_bytes_2,stream>>> (work_buffer, outdat_tscr, nblocks, ndat_proc, ichan);
          if (dsp::Operation::record_time || dsp::Operation::verbose)
            if (stream)
              check_error_stream ("CUDA::SKComputerEngine::compute reduce_sqld [second]", stream);
            else
              check_error ("CUDA::SKComputerEngine::compute reduce_sqld [second]");

          // caculate SK estimator for each block in place [out of place]
          calc_sk_estimate<<<1,nblocks,0,stream>>> (work_buffer, outdat, M_fac, M, nchan*npol);
          if (dsp::Operation::record_time || dsp::Operation::verbose)
            if (stream)
              check_error_stream ("CUDA::SKComputerEngine::compute sk_estimate", stream);
            else
              check_error ("CUDA::SKComputerEngine::compute sk_estimate");

          outdat ++;
          outdat_tscr ++;
        }
      }
#endif

      // now calculate the SK limit for the tscrunched data
      break;
    }

    case dsp::TimeSeries::OrderTFP:
    {
      throw Error (InvalidState, "CUDA::SKComputerEngine::compute",
                   "OrderTFP is unsupported input order");
    }

    default:
    {
      throw Error (InvalidState, "CUDA::SKComputerEngine::compute",
                   "unsupported input order");
    }
  }
}


__global__ void copy1sample ( const float * in_base,
           float2 * out_base,
           uint64_t out_stride,
           uint64_t ndat,
           unsigned M)
{
  const unsigned idat  = blockIdx.x * blockDim.x + threadIdx.x;
  if (idat >= ndat)
    return;

  const unsigned ipol  = blockIdx.z;
  const unsigned ichan = blockIdx.y;
  const unsigned isk   = idat / M;

  const unsigned nchan = gridDim.y;
  const unsigned npol = gridDim.z;

  // forward pointer to pol0 for this chan
  out_base += (ichan * npol + ipol) * out_stride;

  // get the SK estimate (TFP order) for this sample/pol
  const float sk = in_base[isk * nchan * npol + ichan*npol + ipol];

  out_base[idat].x = sk;
  out_base[idat].y = sk;
}



void CUDA::SKComputerEngine::insertsk (const dsp::TimeSeries* input, 
                                       dsp::TimeSeries* output,
                                       unsigned M)
{ 
  // copy the SK estimates to the output timesseries
  if (dsp::Operation::verbose)
    cerr << "CUDA::SKMaskerEngine::insertsk M=" << M << endl;

  uint64_t ndat  = output->get_ndat();
  unsigned nchan = output->get_nchan();
  unsigned npol  = output->get_npol();

  // order is FPT
  const float * in_base = (float *) input->get_dattfp ();
  float2 * out_base     = (float2 *) output->get_datptr (0, 0);

  uint64_t out_stride;
  if (npol == 1)
  {
    out_stride = output->get_datptr (1, 0) - output->get_datptr (0, 0);
  }
  else
  {
    out_stride = output->get_datptr (0, 1) - output->get_datptr (0, 0);
  }

  out_stride /= 2;

  unsigned threads = max_threads_per_block;
  dim3 blocks (ndat / threads, nchan, npol);
  if (ndat % threads)
    blocks.x++;

  cerr << "CUDA::SKComputerEngine::insertsk ndat=" << ndat << " nchan=" << nchan << " npol=" << npol << endl;
  cerr << "CUDA::SKComputerEngine::insertsk out_stride=" << out_stride << endl;
  cerr << "CUDA::SKComputerEngine::insertsk blocks=(" << blocks.x << ", " << blocks.y << ") threads=" << threads << endl;

  copy1sample<<<blocks,threads,0,stream>>> (in_base, out_base, out_stride, ndat, M);

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error( "CUDA::SKComputerEngine::insertsk" );
}
